#include "hip/hip_runtime.h"
#include "connection_gradients.cuh"

__global__ void cud_dense_gradient_calculation(
	data_t* activations, size_t activations_start,
	data_t* gradients, size_t gradients_start, size_t layer_gradients_start, size_t* neuron_gradients_starts,
	data_t* costs, size_t costs_start,
	size_t previous_layer_activations_start, size_t previous_layer_length,
	field_t* weights
)
{
	size_t tid = get_tid();
	if (tid >= previous_layer_length)
		return;

	// Input gradient is bias gradient
	size_t input_gradient_i = gradients_start + layer_gradients_start + neuron_gradients_starts[blockIdx.y];
	data_t input_gradient = gradients[input_gradient_i];
	size_t weight_gradient_i = input_gradient_i + tid + 1;
	field_t weight = weights[tid];
	data_t activation = activations[activations_start + previous_layer_activations_start + tid];
	gradients[weight_gradient_i] = input_gradient * activation;
	atomicAdd(costs + costs_start + previous_layer_activations_start + tid, -input_gradient * weight);
}

__global__ void cud_NEAT_gradient_calculation(
	data_t* activations, size_t activations_start,
	data_t* gradients, size_t gradients_start, size_t layer_gradients_start, size_t* neuron_gradients_starts,
	data_t* costs, size_t costs_start,
	size_t neuron_i, size_t connection_count, field_t* weights, size_t* connection_points, size_t connections_start
)
{
	size_t tid = get_tid();
	if (tid >= connection_count)
		return;

	size_t input_gradient_i = gradients_start + layer_gradients_start + neuron_gradients_starts[neuron_i];
	size_t weight_gradient_i = input_gradient_i + tid + 1;
	size_t connection_input_i = connection_points[connections_start + tid];

	data_t input_gradient = gradients[input_gradient_i];
	gradients[weight_gradient_i] = input_gradient * activations[activations_start + connection_input_i];
	atomicAdd(costs + costs_start + connection_input_i, -input_gradient * weights[connections_start + tid]);
}

__global__ void bias_gradient_subtraction(
	data_t* gradients, size_t gradients_start, size_t layer_gradients_start, size_t* neuron_gradients_starts,
	field_t* biases, size_t layer_length, data_t learning_rate, short* dropout, data_t max_subtracted_gradient
)
{
	size_t tid = get_tid();
	if (tid >= layer_length) return;

	size_t gradient_i = gradients_start + layer_gradients_start + neuron_gradients_starts[tid];
	data_t gradient = gradients[gradient_i];
	biases[tid] -= device_closest_to_zero(max_subtracted_gradient * (-1 + 2 * (gradient >= 0 && max_subtracted_gradient >= 0)), gradient * learning_rate * dropout[tid]);
}

__global__ void cud_dense_gradient_subtraction(
	data_t* gradients, size_t gradients_start, size_t layer_gradients_start, size_t* neuron_gradients_starts,
	field_t* weights, size_t previous_layer_length,
	data_t learning_rate, short* dropout, data_t max_subtracted_gradient
)
{
	size_t tid = get_tid();
	if (tid >= previous_layer_length) return;

	size_t gradient_i = gradients_start + layer_gradients_start + neuron_gradients_starts[blockIdx.y] + tid + 1;
	data_t gradient = gradients[gradient_i];
	size_t weight_i = previous_layer_length * blockIdx.y + tid;
	atomicAdd(weights + weight_i, -device_closest_to_zero(max_subtracted_gradient * (-1 + 2 * (gradient >= 0 && max_subtracted_gradient >= 0)), gradient * learning_rate * dropout[blockIdx.y]));
}

__global__ void cud_NEAT_gradient_subtraction(
	data_t* gradients, size_t gradients_start, size_t layer_gradients_start, size_t* neuron_gradients_starts,
	size_t neuron_i, size_t connection_count, field_t* weights, size_t connections_start,
	data_t learning_rate, short* dropout, data_t max_subtracted_gradient
)
{
	size_t tid = get_tid();
	if (tid >= connection_count) return;

	size_t gradient_i = gradients_start + layer_gradients_start + neuron_gradients_starts[neuron_i] + tid + 1;
	data_t gradient = gradients[gradient_i];
	size_t weight_i = connections_start + tid;
	atomicAdd(weights + weight_i, -device_closest_to_zero(max_subtracted_gradient * (-1 + 2 * (gradient >= 0 && max_subtracted_gradient >= 0)), gradient * learning_rate * dropout[neuron_i]));
}
