#include "hip/hip_runtime.h"
#include "costs.cuh"

__global__ void MSE_derivative(
	data_t* activations, size_t neuron_count, size_t activations_start, size_t last_layer_activations_start,
	data_t* costs, size_t costs_start,
	data_t* Y_hat, size_t output_length
)
{
	size_t tid = get_tid();
	if (tid >= output_length) return;
	size_t t = blockIdx.y;

	data_t predicted = activations[activations_start + neuron_count * t + last_layer_activations_start + tid];
	data_t Y = Y_hat[output_length * t + tid]; 
	//data_t derivative = -2 * (Y_hat[output_length * t + tid] - activations[activations_start + neuron_count * t + last_layer_activations_start + tid]);
	data_t derivative = 2 * (predicted - Y);
	costs[costs_start + t * neuron_count + last_layer_activations_start + tid] = derivative;
}

__global__ void MSE_cost(
	data_t* activations, size_t neuron_count, size_t activations_start, size_t last_layer_activations_start,
	data_t* Y_hat, size_t output_length,
	data_t* cost_write
)
{
	size_t tid = get_tid();
	if (tid >= output_length) return;
	size_t t = blockIdx.y;

	data_t predicted = activations[activations_start + neuron_count * t + last_layer_activations_start + tid];
	data_t Y = Y_hat[output_length * t + tid];
	data_t error = Y - predicted;
	error *= error;
	atomicAdd(cost_write, error);
}

__global__ void log_likelyhood_cost(
	data_t* activations, size_t neuron_count, size_t activations_start, size_t last_layer_activations_start,
	data_t* rewards, size_t output_length,
	data_t* cost
)
{
	size_t tid = get_tid();
	if (tid >= output_length) return;
	size_t t = blockIdx.y;
	
	data_t reward = rewards[t];
	data_t prediction = activations[activations_start + neuron_count * t + last_layer_activations_start + tid];
	data_t output = -log(prediction) * reward;
	
	atomicAdd(cost, output);
}

__global__ void log_likelyhood_derivative(
	data_t* activations, size_t activations_start,
	size_t neuron_count, size_t last_layer_activations_start, size_t output_length,
	data_t* costs, size_t costs_start,
	data_t* rewards
)
{
	size_t tid = get_tid();
	if (tid >= output_length) return;
	size_t t = blockIdx.y;

	data_t reward = rewards[t];
	data_t activation = neuron_count * t + last_layer_activations_start + tid;
	data_t cost_derivative = -(reward / activation);


	size_t cost_write = costs_start + neuron_count * t + last_layer_activations_start + tid;
	costs[cost_write] = cost_derivative;
}

__global__ void PPO_cost(
	data_t* activations, size_t activations_start,
	size_t neuron_count, size_t last_layer_activations_start, size_t output_length,
	data_t* costs, size_t costs_start,
	data_t* rewards
)
{
	size_t tid = get_tid();
	if (tid >= output_length) return;
	size_t t = blockIdx.y;

	data_t ratio = 1;
	if (t) ratio = 
		activations[activations_start + neuron_count * t + last_layer_activations_start + tid] / 
		activations[activations_start + neuron_count * (t - 1) + last_layer_activations_start + tid];

	data_t reward = rewards[t];

	data_t clip = device_clip(ratio, 1 + .2, 1 - .2);
	data_t loss = device_min(ratio * reward, clip * reward);
	
	size_t cost_write = costs_start + neuron_count * t + last_layer_activations_start + tid;
	costs[cost_write] = loss;
}
