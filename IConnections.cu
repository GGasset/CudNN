#include "IConnections.h"

void IConnections::generate_random_values(float** pointer, size_t float_count, size_t start_i)
{
	hiprandGenerator_t generator;
	hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_XORWOW);
	//hiprandSetPseudoRandomGeneratorSeed(generator, 15);
	hiprandGenerateUniform(generator, *pointer + start_i, float_count);
}

void IConnections::mutate_fields(evolution_metadata evolution_values)
{
	float* arr0 = 0;
	hipMalloc(&arr0, sizeof(float) * neuron_count * 3);
	hipDeviceSynchronize();
	generate_random_values(&arr0, neuron_count * 3);
	hipDeviceSynchronize();

	mutate_field_array kernel(neuron_count / 32 + (neuron_count % 32 > 0), 32) (
		biases, neuron_count,
		evolution_values.field_max_evolution, evolution_values.field_mutation_chance,
		arr0
	);
	hipFree(arr0);
	hipDeviceSynchronize();

	hipMalloc(&arr0, sizeof(float) * connection_count * 3);
	hipDeviceSynchronize();
	generate_random_values(&arr0, connection_count * 3);
	hipDeviceSynchronize();

	mutate_field_array kernel(connection_count / 32 + (connection_count % 32 > 0), 32) (
		weights, connection_count,
		evolution_values.field_max_evolution, evolution_values.field_mutation_chance,
		arr0
	);

	hipFree(arr0);
	hipDeviceSynchronize();
}

void IConnections::add_neuron(size_t previous_layer_length, size_t previous_layer_activations_start, float previous_layer_connection_probability, size_t min_connections)
{
}

void IConnections::remove_neuron(size_t neuron_i)
{
}

void IConnections::adjust_to_removed_neuron(size_t neuron_i, std::vector<size_t>* removed_connections_neuron_i)
{
}

void IConnections::adjust_to_added_neuron(size_t added_neuron_i, float connection_probability, std::vector<size_t>* added_connections_neuron_i)
{
}

void IConnections::deallocate()
{
	hipFree(weights);
	hipFree(biases);
	specific_deallocate();
	hipDeviceSynchronize();
}

void IConnections::specific_deallocate()
{
}
