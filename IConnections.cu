#include "IConnections.h"

void IConnections::mutate_fields(evolution_metadata evolution_values)
{
	float* arr0 = 0;
	hipMalloc(&arr0, sizeof(float) * neuron_count * 3);
	hipDeviceSynchronize();
	generate_random_values(&arr0, neuron_count * 3);
	hipDeviceSynchronize();

	mutate_field_array kernel(neuron_count / 32 + (neuron_count % 32 > 0), 32) (
		biases, neuron_count,
		evolution_values.field_max_evolution, evolution_values.field_mutation_chance,
		arr0
	);
	hipFree(arr0);
	hipDeviceSynchronize();

	hipMalloc(&arr0, sizeof(float) * connection_count * 3);
	hipDeviceSynchronize();
	generate_random_values(&arr0, connection_count * 3);
	hipDeviceSynchronize();

	mutate_field_array kernel(connection_count / 32 + (connection_count % 32 > 0), 32) (
		weights, connection_count,
		evolution_values.field_max_evolution, evolution_values.field_mutation_chance,
		arr0
	);

	hipFree(arr0);
	hipDeviceSynchronize();
}

void IConnections::add_neuron(size_t previous_layer_length, size_t previous_layer_activations_start, float previous_layer_connection_probability, size_t min_connections)
{
}

void IConnections::remove_neuron(size_t neuron_i)
{
}

void IConnections::adjust_to_removed_neuron(size_t neuron_i, std::vector<size_t>* removed_connections_neuron_i)
{
}

void IConnections::adjust_to_added_neuron(size_t added_neuron_i, float connection_probability, std::vector<size_t>* added_connections_neuron_i)
{
}

void IConnections::IConnections_clone(IConnections* base)
{
	hipMalloc(&base->weights, sizeof(field_t) * connection_count);
	hipMalloc(&base->biases, sizeof(field_t) * neuron_count);
	hipDeviceSynchronize();
	hipMemcpy(base->weights, weights, sizeof(field_t) * connection_count, hipMemcpyDeviceToDevice);
	hipMemcpy(base->biases, biases, sizeof(field_t) * neuron_count, hipMemcpyDeviceToDevice);
	hipDeviceSynchronize();
	base->neuron_count = neuron_count;
	base->connection_count = connection_count;
	base->contains_irregular_connections = contains_irregular_connections;
}

void IConnections::save(FILE* file)
{
	fwrite(&neuron_count, sizeof(size_t), 1, file);
	fwrite(&connection_count, sizeof(size_t), 1, file);
	fwrite(&contains_irregular_connections, sizeof(unsigned char), 1, file);

	specific_save(file);

	field_t* host_weights = new field_t[connection_count];
	field_t* host_biases = new field_t[neuron_count];

	hipMemcpy(host_weights, weights, sizeof(field_t) * connection_count, hipMemcpyDeviceToHost);
	hipMemcpy(host_biases, biases, sizeof(field_t) * neuron_count, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	fwrite(host_weights, sizeof(field_t), connection_count, file);
	fwrite(host_biases, sizeof(field_t), neuron_count, file);
	delete[] host_weights;
	delete[] host_biases;

}


void IConnections::load_neuron_metadata(FILE* file)
{
	fread(&neuron_count, sizeof(size_t), 1, file);
	fread(&connection_count, sizeof(size_t), 1, file);
	fread(&contains_irregular_connections, sizeof(unsigned char), 1, file);
}

void IConnections::load_IConnections_data(FILE* file)
{
	field_t* host_weights = new field_t[connection_count];
	field_t* host_biases = new field_t[neuron_count];

	fread(host_weights, sizeof(field_t), connection_count, file);
	fread(host_biases, sizeof(field_t), neuron_count, file);

	hipMalloc(&weights, sizeof(field_t) * connection_count);
	hipMalloc(&biases, sizeof(field_t) * neuron_count);
	hipDeviceSynchronize();

	hipMemcpy(weights, host_weights, sizeof(field_t) * connection_count, hipMemcpyHostToDevice);
	hipMemcpy(biases, host_biases, sizeof(field_t) * neuron_count, hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	delete[] host_weights;
	delete[] host_biases;
}

void IConnections::deallocate()
{
	hipFree(weights);
	hipFree(biases);
	specific_deallocate();
	hipDeviceSynchronize();
}

void IConnections::specific_deallocate()
{
}
