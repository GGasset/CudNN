#include "hip/hip_runtime.h"
#include "costs.cuh"

__global__ void MSE_derivative(
	data_t* activations, size_t neuron_count, size_t activations_start, size_t last_layer_activations_start,
	data_t* costs, size_t costs_start,
	data_t* Y_hat, size_t output_length
)
{
	size_t tid = get_tid();
	if (tid >= output_length) return;
	size_t t = blockIdx.y;

	data_t derivative = -2 * (Y_hat[output_length * t + tid] - activations[activations_start + neuron_count * t + last_layer_activations_start + tid]);
	costs[costs_start + t * neuron_count + last_layer_activations_start + tid] = derivative;
}
