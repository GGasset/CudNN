#include "IConnections.h"

void IConnections::generate_random_values(float** pointer, size_t float_count, size_t start_i, float value_divider)
{
	hiprandGenerator_t generator;
	hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_XORWOW);
	hiprandSetPseudoRandomGeneratorSeed(generator, 13);
	hiprandGenerateUniform(generator, *pointer + start_i, float_count);
	multiply_array kernel(float_count / 32 + (float_count % 32 > 0), 32) (
		*pointer + start_i, float_count, 1 / value_divider
	);
	hipDeviceSynchronize();
	hiprandDestroyGenerator(generator);
}

void IConnections::mutate_fields(evolution_metadata evolution_values)
{
	float* arr0 = 0;
	hipMalloc(&arr0, sizeof(float) * neuron_count * 3);
	hipDeviceSynchronize();
	generate_random_values(&arr0, neuron_count * 3);
	hipDeviceSynchronize();

	mutate_field_array kernel(neuron_count / 32 + (neuron_count % 32 > 0), 32) (
		biases, neuron_count,
		evolution_values.field_max_evolution, evolution_values.field_mutation_chance,
		arr0
	);
	hipFree(arr0);
	hipDeviceSynchronize();

	hipMalloc(&arr0, sizeof(float) * connection_count * 3);
	hipDeviceSynchronize();
	generate_random_values(&arr0, connection_count * 3);
	hipDeviceSynchronize();

	mutate_field_array kernel(connection_count / 32 + (connection_count % 32 > 0), 32) (
		weights, connection_count,
		evolution_values.field_max_evolution, evolution_values.field_mutation_chance,
		arr0
	);

	hipFree(arr0);
	hipDeviceSynchronize();
}

void IConnections::add_neuron(size_t previous_layer_length, size_t previous_layer_activations_start, float previous_layer_connection_probability, size_t min_connections)
{
}

void IConnections::remove_neuron(size_t neuron_i)
{
}

void IConnections::adjust_to_removed_neuron(size_t neuron_i, std::vector<size_t>* removed_connections_neuron_i)
{
}

void IConnections::adjust_to_added_neuron(size_t added_neuron_i, float connection_probability, std::vector<size_t>* added_connections_neuron_i)
{
}

void IConnections::IConnections_clone(IConnections* base)
{
	hipMalloc(&base->weights, sizeof(field_t) * connection_count);
	hipMalloc(&base->biases, sizeof(field_t) * neuron_count);
	hipDeviceSynchronize();
	hipMemcpy(base->weights, weights, sizeof(field_t) * connection_count, hipMemcpyDeviceToDevice);
	hipMemcpy(base->biases, biases, sizeof(field_t) * neuron_count, hipMemcpyDeviceToDevice);
	hipDeviceSynchronize();
	base->neuron_count = neuron_count;
	base->connection_count = connection_count;
	base->contains_irregular_connections = contains_irregular_connections;
}

void IConnections::save(FILE* file)
{
	fwrite(&neuron_count, sizeof(size_t), 1, file);
	fwrite(&connection_count, sizeof(size_t), 1, file);
	fwrite(&contains_irregular_connections, sizeof(unsigned char), 1, file);

	specific_save(file);

	field_t* host_weights = new field_t[connection_count];
	field_t* host_biases = new field_t[neuron_count];

	hipMemcpy(host_weights, weights, sizeof(field_t) * connection_count, hipMemcpyDeviceToHost);
	hipMemcpy(host_biases, biases, sizeof(field_t) * neuron_count, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	fwrite(host_weights, sizeof(field_t), connection_count, file);
	fwrite(host_biases, sizeof(field_t), neuron_count, file);
	delete[] host_weights;
	delete[] host_biases;

}

void IConnections::deallocate()
{
	hipFree(weights);
	hipFree(biases);
	specific_deallocate();
	hipDeviceSynchronize();
}

void IConnections::specific_deallocate()
{
}
