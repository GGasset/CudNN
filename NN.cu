#include "hip/hip_runtime.h"
﻿#ifndef NN_DEFINITIONS
#define NN_DEFINITIONS

#include "NN.h"

size_t NN::get_input_length()
{
	return input_length; 
}

size_t NN::get_output_length()
{
	return output_length;
}

NN::NN(ILayer** layers, size_t input_length, size_t layer_count)
{
	this->layers = layers;
	this->input_length = input_length;
	this->layer_count = layer_count;
	set_fields();
}

NN::NN()
{
}

NN::~NN()
{
	deallocate();
}

void NN::set_fields()
{
	output_length = layers[layer_count - 1]->get_neuron_count();

	size_t neuron_count = input_length;
	size_t execution_value_count = 0;
	size_t derivative_count = 0;
	size_t gradient_count = 0;
	contains_recurrent_layers = false;
	for (size_t i = 0; i < layer_count; i++)
	{
		ILayer* layer = layers[i];
		
		contains_recurrent_layers = contains_recurrent_layers || layer->is_recurrent;

		layer->layer_activations_start = neuron_count;
		neuron_count += layer->get_neuron_count();

		layer->execution_values_layer_start = execution_value_count;
		execution_value_count += layer->execution_values_per_neuron * layer->get_neuron_count();

		layer->layer_derivatives_start = derivative_count;
		derivative_count += layer->layer_derivative_count;

			layer->layer_gradients_start = gradient_count;
			gradient_count += layer->layer_gradient_count;
		}
		this->neuron_count = neuron_count;
	output_activations_start = &(layers[layer_count - 1]->layer_activations_start);
	this->execution_value_count = execution_value_count;
	this->derivative_count = derivative_count;
	this->gradient_count = gradient_count;
}

void NN::execute(data_t* input, data_t* execution_values, data_t *activations, size_t t, data_t* output_start_pointer, short copy_output_to_host = true)
{
	hipMemcpy(activations + t * neuron_count, input + input_length * t, sizeof(data_t) * input_length, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	for (size_t i = 0; i < layer_count; i++)
	{
		layers[i]->execute(activations, neuron_count * t, execution_values, execution_value_count * t);
		hipDeviceSynchronize();
	}
	if (copy_output_to_host)
	{
		hipMemcpy(output_start_pointer + output_length * t, activations + neuron_count * t + *output_activations_start, sizeof(data_t) * output_length, hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
	}
}

void NN::set_up_execution_arrays(data_t** execution_values, data_t** activations, size_t t_count)
{
	hipMalloc(execution_values, sizeof(data_t) * execution_value_count * t_count);
	hipMalloc(activations, sizeof(data_t) * neuron_count * t_count);
	hipDeviceSynchronize();
	hipMemset(*execution_values, 0, sizeof(data_t) * execution_value_count * t_count);
	hipMemset(*activations, 0, sizeof(data_t) * neuron_count * t_count);
	hipDeviceSynchronize();
}

data_t* NN::batch_execute(data_t* input, size_t t_count)
{
	data_t* execution_values = 0;
	data_t* activations = 0;
	set_up_execution_arrays(&execution_values, &activations, t_count);

	data_t* outputs = new data_t[output_length * t_count];
	for (size_t i = 0; i < output_length * t_count; i++)
	{
		outputs[i] = 0;
	}
	for (size_t t = 0; t < t_count; t++)
	{
		execute(input, execution_values, activations, t, outputs, 1);
	}


	hipFree(execution_values);
	hipFree(activations);
	hipDeviceSynchronize();
	return outputs;
}

data_t* NN::inference_execute(data_t* input)
{
	return batch_execute(input, 1);
}


data_t NN::adjust_learning_rate(
	data_t learning_rate,
	data_t cost,
	LearningRateAdjusters adjuster,
	data_t max_learning_rate,
	data_t previous_cost
)
{
	data_t new_learning_rate = learning_rate;
	if (adjuster == LearningRateAdjusters::none) return new_learning_rate;
	if (previous_cost != 0 && cost != 0)
		switch (adjuster) {
			case LearningRateAdjusters::high_learning_high_learning_rate:
				{
					data_t learning = previous_cost / cost;
					new_learning_rate += learning;
				}
				break;
			case LearningRateAdjusters::high_learning_low_learning_rate:
				{
					data_t learning = previous_cost / cost;
					new_learning_rate -= learning;
					new_learning_rate = max<data_t>(0, new_learning_rate);
				}
				break;
			default:
				break;
		}
	switch (adjuster) {
		case LearningRateAdjusters::cost_times_learning_rate:
			new_learning_rate = learning_rate * cost;
			break;
		default:
			break;
	}
	return min(new_learning_rate, max_learning_rate);

}

data_t NN::calculate_output_costs(
	CostFunctions cost_function,
	size_t t_count,
	data_t* Y_hat,
	data_t* activations, size_t activations_start,
	data_t* costs, size_t costs_start
)
{
	data_t* cost = 0;
	hipMalloc(&cost, sizeof(data_t));
	hipDeviceSynchronize();
	hipMemset(cost, 0, sizeof(data_t));
	hipDeviceSynchronize();
	switch (cost_function)
	{
	case CostFunctions::MSE:
		MSE_derivative kernel(dim3(output_length / 32 + (output_length % 32 > 0), t_count), 32) (
			activations, neuron_count, activations_start, *output_activations_start,
			costs, costs_start,
			Y_hat, output_length
		);
		MSE_cost kernel(dim3(output_length / 32 + (output_length % 32 > 0), t_count), 32) (
			activations, neuron_count, activations_start, *output_activations_start,
			Y_hat, output_length,
			cost
		);
		break;
	case CostFunctions::log_likelyhood:
		log_likelyhood_derivative kernel(dim3(output_length / 32 + (output_length % 32 > 0), t_count), 32) (
			activations, activations_start,
			neuron_count, *output_activations_start, output_length,
			costs, costs_start,
			Y_hat
		);
		log_likelyhood_cost kernel(dim3(output_length / 32 + (output_length % 32 > 0), t_count), 32) (
			activations, neuron_count, activations_start, *output_activations_start,
			Y_hat, output_length,
			cost
		);
		break;
	case CostFunctions::PPO:
		PPO_cost kernel(dim3(output_length / 32 + (output_length % 32 > 0), t_count), 32) (
			activations, activations_start, 
			neuron_count, *output_activations_start, output_length,
			costs, costs_start,
			Y_hat
		);
		break;
	default:
		break;
	}
	hipDeviceSynchronize();
	multiply_array kernel(1, 1) (
		cost, 1, 1.0 / (output_length * t_count)
	);
	data_t host_cost = 0;
	hipDeviceSynchronize();
	hipMemcpy(&host_cost, cost, sizeof(data_t), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipFree(cost);
	return host_cost;
}

void NN::training_execute(
	size_t t_count,
	data_t* X,
	data_t** Y,
	bool copy_Y_to_host,
	data_t** execution_values,
	data_t** activations,
	size_t arrays_t_length
)
{
	data_t* prev_execution_values = 0;
	data_t* prev_activations = 0;
	if (arrays_t_length)
	{
		prev_execution_values = *execution_values;
		prev_activations = *activations;
	}
	set_up_execution_arrays(execution_values, activations, t_count + arrays_t_length);
	if (arrays_t_length)
	{
		hipMemcpy(*execution_values, prev_execution_values, sizeof(data_t) * execution_value_count * arrays_t_length, hipMemcpyDeviceToDevice);
		hipMemcpy(*activations, prev_activations, sizeof(data_t) * neuron_count * arrays_t_length, hipMemcpyDeviceToDevice);
		hipDeviceSynchronize();
		hipFree(prev_execution_values);
		hipFree(prev_activations);
	}


	if (copy_Y_to_host)
	{
		*Y = new data_t[output_length * t_count];
		for (size_t i = 0; i < output_length * t_count; i++)
		{
			(*Y)[i] = 0;
		}
	}

	for (size_t t = 0; t < t_count; t++)
	{
		execute(X, (*execution_values) + execution_value_count * arrays_t_length, (*activations) + neuron_count * arrays_t_length, t, copy_Y_to_host ? *Y : 0, copy_Y_to_host);
	}
}


data_t NN::train(
	size_t t_count,
	data_t* execution_values,
	data_t* activations,
	data_t* Y_hat,
	bool is_Y_hat_on_host_memory,
	size_t Y_hat_value_count,
	CostFunctions cost_function,
	data_t learning_rate,
	data_t gradient_clip,
	float dropout_rate
)
{
	data_t* costs = 0;
	hipMalloc(&costs, sizeof(data_t) * neuron_count * t_count);
	hipDeviceSynchronize();

	hipMemset(costs, 0, sizeof(data_t) * neuron_count * t_count);
	hipDeviceSynchronize();
	
	if (is_Y_hat_on_host_memory)
	{
		data_t* temp_Y_hat = 0;
		hipMalloc(&temp_Y_hat, sizeof(data_t) * Y_hat_value_count);
		hipMemcpy(temp_Y_hat, Y_hat, sizeof(data_t) * Y_hat_value_count, hipMemcpyHostToDevice);
		Y_hat = temp_Y_hat;
	}
	data_t cost = calculate_output_costs(cost_function, t_count, Y_hat, activations, 0, costs, 0);
	hipDeviceSynchronize();

	data_t* gradients = 0;
	backpropagate(
		t_count, costs, activations, execution_values, &gradients
	);

	for (size_t t = 0; t < t_count; t++)
	{
		subtract_gradients(gradients, gradient_count * t, learning_rate, dropout_rate, gradient_clip);
	}

	if (is_Y_hat_on_host_memory) hipFree(Y_hat);
	hipFree(activations);
	hipFree(execution_values);
	hipFree(costs);
	hipFree(gradients);
	hipDeviceSynchronize();

	return cost;
}

data_t NN::training_batch(
	size_t t_count,
	data_t* X,
	data_t* Y_hat,
	bool is_Y_hat_on_host_memory,
	size_t Y_hat_value_count,
	CostFunctions cost_function,
	data_t learning_rate,
	data_t** Y,
	bool copy_Y_to_host,
	data_t gradient_clip,
	float dropout_rate
)
{
	data_t* execution_values = 0;
	data_t* activations = 0;
	training_execute(
		t_count,
		X,
		Y,
		copy_Y_to_host,
		&execution_values,
		&activations
	);
	return train(
		t_count, 
		execution_values,
		activations,
		Y_hat,
		is_Y_hat_on_host_memory,
		Y_hat_value_count,
		cost_function,
		learning_rate,
		gradient_clip,
		dropout_rate
	);
}

void NN::backpropagate(
	size_t t_count, 
	data_t* costs,
	data_t* activations, 
	data_t* execution_values,
	data_t** gradients
)
{
	data_t* derivatives = 0;
	if (!*gradients)
	{
		hipMalloc(gradients, sizeof(data_t) * t_count * gradient_count);
		hipMemset(*gradients, 0, sizeof(data_t) * t_count * gradient_count);
	}
	if (derivative_count)
	{
		hipMalloc(&derivatives, sizeof(data_t) * t_count * derivative_count);
		hipMemset(derivatives, 0, sizeof(data_t) * t_count * derivative_count);
	}
	hipDeviceSynchronize();

	size_t activations_start = 0;
	size_t execution_values_start = 0;
	size_t derivatives_start = 0;
	size_t gradients_start = 0;
	for (size_t t = 0; t < t_count; t++)
	{
		activations_start = neuron_count * t;
		derivatives_start = derivative_count * t;
		execution_values_start = execution_value_count * t;
		calculate_derivatives(
			activations, activations_start, 
			derivatives, derivatives_start - derivative_count, derivatives_start,
			execution_values, execution_values_start
		);
	}
	for (int t = t_count - 1; t >= 0; t--)
	{
		gradients_start = gradient_count * t;
		size_t next_gradient_start = gradients_start + gradient_count;
		next_gradient_start -= next_gradient_start * (t == t_count - 1);

		derivatives_start = derivative_count * t;
		activations_start = neuron_count * t;

		calculate_gradients(
			activations, activations_start,
			execution_values, execution_values_start,
			costs, activations_start,
			*gradients, gradients_start, next_gradient_start,
			derivatives, derivatives_start, derivatives_start - derivative_count
		);
	}

	if (!stateful && contains_recurrent_layers)
		delete_memory();
	if (derivative_count) hipFree(derivatives);
}

void NN::calculate_derivatives(
	data_t* activations, size_t activations_start,
	data_t* derivatives, size_t previous_derivatives_start, size_t derivatives_start,
	data_t* execution_values, size_t execution_values_start
)
{
	// Todo: make layer gradient calculation async
	for (size_t i = 0; i < layer_count; i++)
	{
		layers[i]->calculate_derivatives(
			activations, activations_start,
			derivatives, previous_derivatives_start, derivatives_start,
			execution_values, execution_values_start
		);
		hipDeviceSynchronize();
	}
}

void NN::calculate_gradients(
	data_t* activations, size_t activations_start,
	data_t* execution_values, size_t execution_values_start,
	data_t* costs, size_t costs_start, 
	data_t* gradients, size_t gradients_start, size_t next_gradients_start, 
	data_t* derivatives, size_t derivatives_start, size_t previous_derivatives_start
)
{
	for (int i = layer_count - 1; i >= 0; i--)
	{
		layers[i]->calculate_gradients(
			activations, activations_start,
			execution_values, execution_values_start,
			derivatives, derivatives_start,
			gradients, next_gradients_start, gradients_start,
			costs, costs_start
		);
		hipDeviceSynchronize();
	}
}

/*data_t* NN::calculate_GAE_advantage(size_t t_count, data_t gamma, data_t lambda, NN* value_function_estimator, data_t* value_function_state, data_t* rewards)
{
	return nullptr;
}*/

data_t *calculate_GAE_advantage(
	size_t t_count,
	data_t gamma, data_t lambda,
	NN *value_function_estimator, data_t *value_function_state, data_t estimator_learning_rate, data_t estimator_gradient_clip, data_t estimator_dropout_rate, bool is_state_on_host, bool free_state,
	data_t *rewards, bool is_reward_on_host, bool free_rewards
)
{
	if (!value_function_estimator) return (0);

	data_t *discounted_rewards = 0;
	hipMalloc(&discounted_rewards, sizeof(data_t) * t_count);
	hipDeviceSynchronize();
	if (!discounted_rewards) return (0);

	hipMemset(discounted_rewards, 0, sizeof(data_t) * t_count);
	hipDeviceSynchronize();

	calculate_discounted_rewards kernel(t_count / 32 + (t_count % 32 > 0), 32) (
		t_count, gamma, rewards, discounted_rewards
	);
	hipDeviceSynchronize();

	data_t *host_value_functions = 0;
	value_function_estimator->training_batch( // TODO Returns a pointer to host memory and then copies it. Slow!
		t_count,
		value_function_state, discounted_rewards, 0, t_count,
		CostFunctions::MSE, estimator_learning_rate,
		&host_value_functions, 1, estimator_gradient_clip, estimator_dropout_rate
	);

	data_t* value_functions = 0;
	hipMalloc(&value_functions, sizeof(data_t) * value_function_estimator->get_output_length() * t_count);
	hipDeviceSynchronize();
	if (!value_functions || !host_value_functions)
		return 0;
	hipMemcpy(value_functions, host_value_functions, sizeof(data_t) * value_function_estimator->get_output_length() * t_count, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	delete[] host_value_functions;

	data_t *deltas = 0;
	hipMalloc(&deltas, sizeof(data_t) * t_count);
	hipDeviceSynchronize();
	if (!deltas) return (0);

	hipMemset(deltas, 0, sizeof(data_t) * t_count);
	hipDeviceSynchronize();

	calculate_deltas kernel(t_count / 32 + (t_count % 32 > 0), 32) (
		t_count, 
		gamma, 
		rewards, 
		value_functions, 
		deltas
	);
	hipDeviceSynchronize();

	data_t* advantages = 0;
	hipMalloc(&advantages, sizeof(data_t) * t_count);
	hipDeviceSynchronize();
	if (!advantages) return 0;

	hipMemset(advantages, 0, sizeof(data_t) * t_count);
	hipDeviceSynchronize();

	parallel_calculate_GAE_advantage kernel(t_count / 32 + (t_count % 32 > 0), 32) (
		t_count,
		gamma, lambda,
		deltas, advantages
	);
	hipDeviceSynchronize();

	hipFree(deltas);
	hipFree(discounted_rewards);
	hipFree(value_functions);
	delete[] value_functions;

	return 0;
}

void NN::subtract_gradients(data_t* gradients, size_t gradients_start, data_t learning_rate, float dropout_rate, data_t gradient_clip)
{
	reset_NaNs kernel(gradient_count / 32 + (gradient_count % 32 > 0), 32) (
		gradients + gradients_start, 0, gradient_count
	);
	hipDeviceSynchronize();
	
	for (size_t i = 0; i < layer_count; i++)
	{
		ILayer* current_layer = layers[i];
		size_t layer_length = current_layer->get_neuron_count();

		short* dropout = 0;
		float* normalized_random_samples = 0;
		hipMalloc(&dropout, sizeof(short) * layer_length);
		hipMalloc(&normalized_random_samples, sizeof(float) * layer_length);
		hipDeviceSynchronize();
		
		hipMemset(dropout, 0, sizeof(short) * layer_length);
		IConnections::generate_random_values(&normalized_random_samples, layer_length, 0, 1);
		hipDeviceSynchronize();
		cud_set_dropout kernel(layer_length / 32 +  (layer_length % 32 > 0), 32) (dropout_rate, normalized_random_samples, dropout, layer_length);
		hipDeviceSynchronize();

		current_layer->subtract_gradients(gradients, gradients_start, learning_rate, dropout, gradient_clip);

		hipFree(dropout);
		hipFree(normalized_random_samples);
		hipDeviceSynchronize();
	}
	hipDeviceSynchronize();
}

void NN::evolve()
{
	for (size_t i = 0; i < layer_count; i++)
	{
		layers[i]->mutate_fields(evolution_values);
		layers[i]->connections->mutate_fields(evolution_values);
	}
	if (evolution_values.layer_addition_probability > get_random_float())
	{
		printf("Adding layer\n");
		NeuronTypes insert_type = (NeuronTypes)(rand() % NeuronTypes::last_neuron_entry);
		size_t insert_i = layer_count > 1 ? rand() % (layer_count - 1) : 0;
		
		size_t previous_layer_length = input_length;
		size_t previous_layer_activations_start = 0;
		if (insert_i)
		{
			ILayer* previous_layer = layers[insert_i];
			previous_layer_length = previous_layer->get_neuron_count();
			previous_layer_activations_start = previous_layer->layer_activations_start;
		}
		
		IConnections* new_connections = new NeatConnections(previous_layer_activations_start, previous_layer_length, 1);
		ILayer* new_layer = 0;

		switch (insert_type)
		{
		case NeuronTypes::Neuron:
			new_layer = new NeuronLayer(new_connections, 1, (ActivationFunctions)(rand() % ActivationFunctions::activations_last_entry));
			break;
		case NeuronTypes::LSTM:
			new_layer = new LSTMLayer(new_connections, 1);
			break;
		default:
			throw "Neuron_type not added to evolve method";
			break;
		}
		add_layer(insert_i, new_layer);
	}
	if (evolution_values.neuron_deletion_probability > get_random_float() && layer_count > 1)
	{
		printf("removing neuron\n");
		size_t layer_i = rand() % (layer_count - 1);
		remove_neuron(layer_i);
	}
	if (evolution_values.neuron_addition_probability > get_random_float() && layer_count > 1)
	{
		printf("adding_neuron\n");
		size_t layer_i = rand() % (layer_count - 1);
		add_neuron(layer_i);
	}
	float* evolution_values_pointer = (float*)(&evolution_values);
	for (size_t i = 0; i < sizeof(evolution_metadata) / sizeof(float); i++)
	{
		evolution_values_pointer[i] +=
			evolution_values.evolution_metadata_field_max_mutation *
			(evolution_values.evolution_metadata_field_mutation_chance > get_random_float()) *
			(1 - 2 * (get_random_float() > .5));
	}
}

void NN::add_layer(size_t insert_i, ILayer* layer)
{
	ILayer** tmp_layers = layers;
	layer_count++;

	// insert layer
	layers = new ILayer * [layer_count];
	for (size_t i = 0; i < insert_i; i++)
		layers[i] = tmp_layers[i];
	layers[insert_i] = layer;
	for (size_t i = insert_i + 1; i < layer_count; i++)
		layers[i] = tmp_layers[i - 1];

	// Update info
	set_fields();
	size_t added_neuron_count = layer->get_neuron_count();
	size_t added_layer_activations_start = layer->layer_activations_start;
	for (size_t i = 0; i < added_neuron_count; i++)
	{
		adjust_to_added_neuron(insert_i, added_layer_activations_start + i);
	}
	set_fields();
}

void NN::add_output_neuron()
{
	add_neuron(layer_count - 1);
}

void NN::add_input_neuron()
{
	for (size_t i = 0; i < layer_count; i++)
	{
		adjust_to_added_neuron(-1, input_length);
	}
	input_length++;
	set_fields();
}

void NN::add_neuron(size_t layer_i)
{

	size_t previous_layer_length = input_length;
	size_t previous_layer_activations_start = 0;
	if (layer_i)
	{
		ILayer *previous_layer = layers[layer_i];
		previous_layer_length = previous_layer->get_neuron_count();
		previous_layer_activations_start = previous_layer->layer_activations_start;
	}
	size_t added_neuron_i = layers[layer_i]->layer_activations_start + layers[layer_i]->get_neuron_count();
	layers[layer_i]->add_neuron(previous_layer_length, previous_layer_activations_start, 1, 0);
	adjust_to_added_neuron(layer_i, added_neuron_i);
	set_fields();
}

void NN::adjust_to_added_neuron(int layer_i, size_t neuron_i)
{
	size_t layer_distance_from_added_neuron = 1;
	for (int i = layer_i + 1; i < layer_count; i++, layer_distance_from_added_neuron++)
	{
		float connection_probability = 1.0 / layer_distance_from_added_neuron;
		connection_probability += (1 - connection_probability) * evolution_values.layer_distance_from_added_neuron_connection_addition_modifier;
		layers[i]->adjust_to_added_neuron(neuron_i, connection_probability);
	}
}

void NN::remove_neuron(size_t layer_i)
{
	if (layers[layer_i]->get_neuron_count() == 1)
		return;
	size_t layer_neuron_count = layers[layer_i]->get_neuron_count();
	remove_neuron(layer_i, rand() % layer_neuron_count);
}

void NN::remove_neuron(size_t layer_i, size_t layer_neuron_i)
{
	size_t removed_neuron_i = layers[layer_i]->layer_activations_start + layer_neuron_i;
	layers[layer_i]->remove_neuron(layer_neuron_i);
	for (size_t i = layer_i + 1; i < layer_count; i++)
		layers[i]->adjust_to_removed_neuron(removed_neuron_i);

	set_fields();
}

void NN::delete_memory()
{
	for (size_t i = 0; i < layer_count; i++)
		layers[i]->delete_memory();
}

NN* NN::clone()
{
	NN* clone = new NN();
	clone->layer_count = layer_count;
	clone->neuron_count = neuron_count;
	clone->input_length = input_length;
	clone->output_length = output_length;
	
	clone->layers = new ILayer*[layer_count];
	for (size_t i = 0; i < layer_count; i++)
	{
		clone->layers[i] = layers[i]->layer_specific_clone();
		layers[i]->ILayerClone(clone->layers[i]);
	}
	clone->set_fields();
	clone->evolution_values = evolution_values;
	clone->contains_recurrent_layers = contains_recurrent_layers;
	return clone;
}

void NN::save(const char *pathname)
{
	FILE *file = fopen(pathname, "wb");
	if (!file)
		return;
	save(file);
	fclose(file);
}

void NN::save(FILE* file)
{
	fwrite(&layer_count, sizeof(size_t), 1, file);
	fwrite(&input_length, sizeof(size_t), 1, file);
	for (size_t i = 0; i < layer_count; i++)
	{
		size_t layer_type = (size_t)layers[i]->layer_type;
		fwrite(&layer_type, sizeof(size_t), 1, file);
	}

	for (size_t i = 0; i < layer_count; i++)
	{
		size_t connection_type = (size_t)layers[i]->connections->connection_type;
		fwrite(&connection_type, sizeof(size_t), 1, file);
	}

	for (size_t i = 0; i < layer_count; i++)
	{
		layers[i]->save(file);
		layers[i]->connections->save(file);
	}
}

NN* NN::load(const char *pathname, bool load_state)
{
	FILE *file = fopen(pathname, "rb");
	if (!file)
		return 0;
	NN *out = load(file);
	fclose(file);

	if (!load_state) out->delete_memory();
	return out;
}

NN* NN::load(FILE* file)
{
	NN* output = new NN();

	fread(&(output->layer_count), sizeof(size_t), 1, file);
	fread(&(output->input_length), sizeof(size_t), 1, file);

	size_t layer_count = output->layer_count;

	NeuronTypes *neuron_types = new NeuronTypes[layer_count];
	ConnectionTypes *connection_types = new ConnectionTypes[layer_count];

	ILayer **output_layers = new ILayer*[layer_count];

	fread(neuron_types, sizeof(NeuronTypes), layer_count, file);
	fread(connection_types, sizeof(ConnectionTypes), layer_count, file);

	for (size_t i = 0; i < layer_count; i++)
	{
		ILayer *layer = 0;
		IConnections *connections = 0;
		switch (neuron_types[i])
		{
			case NeuronTypes::Neuron:
				layer = new NeuronLayer();
				break;
			case NeuronTypes::LSTM:
				layer = new LSTMLayer();
				break;
			default:
				break;
		}
		switch (connection_types[i])
		{
			case ConnectionTypes::Dense:
				connections = new DenseConnections();
				break;
			case ConnectionTypes::NEAT:
				connections = new NeatConnections();
				break;
			default:
				break;
		}
		layer->load(file);
		connections->load(file);

		layer->connections = connections;
		output_layers[i] = layer;
	}

	delete[] connection_types;
	delete[] neuron_types;

	output->layers = output_layers;
	output->set_fields();
	return output;
}

void NN::deallocate()
{
	for (size_t i = 0; i < layer_count; i++)
	{
		layers[i]->deallocate();
		delete layers[i];
	}
	delete[] layers;
}

void NN::print_shape()
{
	printf("%i ", input_length);
	for (size_t i = 0; i < layer_count; i++)
		printf("%i ", layers[i]->get_neuron_count());
	printf("\n");
}


#endif
