#include "hip/hip_runtime.h"
#ifndef CUDA_LINEAR_FUNCTIONS
#define CUDA_LINEAR_FUNCTIONS

#include "hip/hip_runtime.h"
#include ""

#include "linear_functions.cuh"

#include "data_type.h"

__global__ void cud_dense_linear_function(
	size_t previous_layer_length, field_t* weights,
	size_t activations_start, size_t previous_layer_activations_start, data_t* activations,
	size_t execution_values_start, size_t execution_values_layer_start, size_t layer_execution_values_per_neuron, data_t* execution_values)
{
	size_t connected_activation_i = activations_start + previous_layer_activations_start + threadIdx.x;
	size_t execution_values_i = execution_values_start + execution_values_layer_start + layer_execution_values_per_neuron * blockIdx.x;

	field_t current_weight = weights[previous_layer_length * blockIdx.x + threadIdx.x];
	execution_values[execution_values_i] += current_weight * activations[connected_activation_i];
}

__global__ void cud_add_biases(
	field_t* biases,
	size_t execution_values_start, size_t execution_values_layer_start, size_t layer_execution_values_per_neuron, data_t* execution_values)
{
	size_t execution_values_i = execution_values_start + execution_values_layer_start + layer_execution_values_per_neuron * threadIdx.x;
	execution_values[execution_values_i] += biases[threadIdx.x];
}

__global__ void cud_dense_linear_function_derivative(
	size_t activations_start, size_t previous_layer_activations_start, size_t previous_layer_length, data_t* activations,
	size_t derivatives_start, size_t derivatives_layer_start, size_t derivatives_per_neuron, data_t* derivatives,
	field_t* weights
)
{
	size_t activation_i = activations_start + previous_layer_activations_start + threadIdx.x;
	size_t weight_i = previous_layer_length * blockIdx.x + threadIdx.x;
	size_t connection_derivative = activations[activation_i] + weights[weight_i];

	size_t write_i = derivatives_start + derivatives_layer_start + derivatives_per_neuron * blockIdx.x;
	derivatives[write_i] += connection_derivative;
}

#endif