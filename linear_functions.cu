#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "linear_functions.cuh"

#include "data_type.h"


__global__ void cud_dense_linear_function(
	size_t previous_layer_length, field_t* weights,
	size_t activations_start, size_t previous_layer_activations_start, data_t* activations,
	size_t execution_values_start, size_t execution_values_layer_start, size_t layer_execution_values_per_neuron, data_t* execution_values)
{
	size_t tid = get_tid();
	if (tid >= previous_layer_length)
		return;
	size_t connected_activation_i = activations_start + previous_layer_activations_start + tid;
	size_t execution_values_i = execution_values_start + execution_values_layer_start + layer_execution_values_per_neuron * blockIdx.y;

	field_t current_weight = weights[previous_layer_length * blockIdx.y + tid];
	atomicAdd(execution_values + execution_values_i, current_weight * activations[connected_activation_i]);
}

__global__ void cud_NEAT_linear_function(
	size_t connection_count, field_t* weights, size_t* connection_points, size_t* connection_neuron_i,
	size_t activations_start, data_t* activations,
	size_t execution_values_start, size_t execution_values_layer_start, size_t layer_execution_values_per_neuron, data_t* execution_values
)
{
	size_t tid = get_tid();
	if (tid >= connection_count)
		return;
	
	size_t neuron_i = connection_neuron_i[tid];
	size_t connection_i = connection_points[tid];
	size_t execution_values_i = execution_values_start + execution_values_layer_start + layer_execution_values_per_neuron * neuron_i;
	atomicAdd(execution_values + execution_values_i, activations[activations_start + connection_i] * weights[tid]);
}

__global__ void cud_add_biases(
	size_t layer_length, field_t* biases,
	size_t execution_values_start, size_t execution_values_layer_start, size_t layer_execution_values_per_neuron, data_t* execution_values)
{
	size_t tid = get_tid();
	if (tid >= layer_length)
		return;
	size_t execution_values_i = execution_values_start + execution_values_layer_start + layer_execution_values_per_neuron * tid;
	atomicAdd(execution_values + execution_values_i, biases[tid]);
}

__global__ void cud_dense_linear_function_derivative(
	size_t activations_start, size_t previous_layer_activations_start, size_t previous_layer_length, data_t* activations,
	size_t derivatives_start, size_t derivatives_layer_start, size_t derivatives_per_neuron, data_t* derivatives,
	field_t* weights
)
{
	size_t tid = get_tid();
	if (tid >= previous_layer_length)
		return;
	size_t activation_i = activations_start + previous_layer_activations_start + tid;
	size_t weight_i = previous_layer_length * blockIdx.y + tid;
	data_t connection_derivative = activations[activation_i] + weights[weight_i];

	size_t write_i = derivatives_start + derivatives_layer_start + derivatives_per_neuron * blockIdx.y;
	atomicAdd(derivatives + write_i, connection_derivative);
}

__global__ void cud_NEAT_linear_function_derivative(
	size_t activations_start, data_t* activations,
	size_t derivatives_start, size_t derivatives_layer_start, size_t derivatives_per_neuron, data_t* derivatives,
	size_t connection_count, field_t* weights, size_t* connection_points, size_t* connection_neuron_i
)
{
	size_t tid = get_tid();
	if (tid >= connection_count)
		return;

	size_t neuron_i = connection_neuron_i[tid];
	size_t activation_i = connection_points[tid];
	size_t write_i = derivatives_start + derivatives_layer_start + derivatives_per_neuron * neuron_i;
	atomicAdd(derivatives + write_i, activations[activation_i] + weights[tid]);
}

__global__ void cud_add_bias_derivative(
	size_t layer_length, 
	size_t derivatives_start, size_t derivatives_layer_start, size_t derivatives_per_neuron, data_t* derivatives
)
{
	size_t tid = get_tid();
	if (tid >= layer_length)
		return;
	atomicAdd(derivatives + derivatives_start + derivatives_layer_start + derivatives_per_neuron * tid, 1);
}
