﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

#include "NN.h"
#include "DenseNeuronLayer.h"

int main()
{
	const size_t input_length = 30;
	const size_t output_length = 30;
	data_t X[input_length];

	const size_t shape_length = 3;
	size_t shape[shape_length]{ input_length, 20, output_length };
	ILayer** layers = new ILayer * [shape_length - 1];

	size_t gradient_count = 0;
	size_t neuron_count = 0;
	for (size_t i = 1; i < shape_length; i++)
	{
		layers[i - 1] = new DenseNeuronLayer(gradient_count, shape[i], neuron_count, shape[i - 1], ActivationFunctions::sigmoid);
		gradient_count += layers[i - 1]->layer_gradient_count;
		neuron_count += shape[i];
	}

	NN n = NN(layers, input_length, shape_length - 1, 0);
	for (size_t i = 0; i < 100; i++)
	{
		for (size_t j = 0; j < input_length; j++)
		{
			X[j] = 1 + rand() % 1000 / 1000.0;
			printf("%f ", X[j]);
		}
		printf("\n\n\n");

		data_t *y = n.execute(X);
		for (size_t j = 0; j < output_length; j++)
		{
			printf("%f  ", y[j]);
		}
		std::cout << "\n\n\n\n\n" << std::endl;
		delete[] y;
	}
	n.deallocate();
}