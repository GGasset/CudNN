#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

#include "NN_constructor.h"

/*
static int increment_i(size_t size, int to_increment, size_t i)
{
	i += to_increment;+
	i -= (i - i % size) * (i >= size);
	i = (i % size) * (i < 0) + i * (i >= 0);
	return i;
}*/

template<typename t>
t abs(t a)
{
	return a * (-1 + 2 * (a > 0));
}

int main()
{
	hipSetDevice(0);

	const size_t input_length = 2;
	const size_t output_length = 2;
	/*const size_t t_count = 5;
	data_t X[input_length * t_count]{};
	data_t Y_hat[output_length * t_count]{};

	for (size_t t = 0; t < t_count; t++)
	{
		for (size_t i = 0; i < input_length; i++)
		{
			X[t * input_length + i] = (.2);//  / t_count * ((t + 1)) + .2 / t_count;
		}
		 for (size_t i = 0; i < output_length; i++)
		{
			Y_hat[t * output_length + i] = .05;//(.2) / t_count * (t + 1) + .2 / t_count + (.2) / t_count * (i + 1) / output_length;
		}
	}*/



	NN* n = NN_constructor()
		//.append_layer(ConnectionTypes::Dense, NeuronTypes::Neuron, 3, ActivationFunctions::sigmoid)
		//.append_layer(ConnectionTypes::Dense, NeuronTypes::Neuron, 4, ActivationFunctions::sigmoid)
		//.append_layer(ConnectionTypes::Dense, NeuronTypes::Neuron, 3, ActivationFunctions::sigmoid)
		.append_layer(ConnectionTypes::Dense, NeuronTypes::Neuron, output_length, ActivationFunctions::sigmoid)
		.construct(input_length);
	n->stateful = true;

	const size_t epochs = 10000;
	data_t epochs_mean_reward[epochs];
	const size_t max_steps = 70;
	for (size_t i = 0; i < epochs; i++)
	{
		// RL demonstration
		data_t x = 0;
		data_t y = 0;

		data_t target_x = rand() % 50 + 50;
		data_t target_y = rand() % 50 + 50;
		target_x *= 1 - 2 * (rand() % 2);
		target_y *= 1 - 2 * (rand() % 2);

		data_t X[2];
		data_t *Y = 0;
		data_t* execution_values = 0;
		data_t* activations = 0;
		
		data_t rewards[max_steps];
		
		bool success = false;
		size_t actual_steps = 0;
		data_t max_reward = 0;
		data_t mean_reward = 0;
		data_t mean_output[output_length] {0, 0};
		for (size_t step_i = 0; step_i < max_steps; step_i++)
		{
			actual_steps++;
			rewards[step_i] = 0;

			data_t target_direction_x = target_x - x;
			data_t target_direction_y = target_y - y;
			data_t target_distance = abs(target_direction_x) + abs(target_direction_y);

			X[0] = target_direction_x / 200 + .25;
			X[1] = target_direction_y / 200 + .25;

			n->training_execute(
				1, X, &Y, true,
				&execution_values, &activations,
				step_i
			);

			x += (Y[0] - .5) * 2 * 3;
			y += (Y[1] - .5) * 2 * 3;

			mean_output[0] += Y[0];
			mean_output[1] += Y[1];


			data_t new_target_direction_x = target_x - x;
			data_t new_target_direction_y = target_y - y;
			data_t new_target_distance = abs(new_target_direction_x) + abs(new_target_direction_y);

			rewards[step_i] += ((target_distance > new_target_distance) - (target_distance <= new_target_distance))/* * (1 / (1 + (new_target_distance / (target_x + target_y)))*/;
			//rewards[step_i] /= Y[0] + Y[1];

			max_reward += (abs(rewards[step_i]) - max_reward) * (abs(rewards[step_i]) > max_reward);
			mean_reward += rewards[step_i];

			if (success = abs(new_target_distance) < 3) break;
		}

		mean_reward /= actual_steps;
		epochs_mean_reward[i] = mean_reward;
		n->train(actual_steps,
			execution_values, activations, 
			rewards, true, actual_steps,
			CostFunctions::log_likelyhood, .03, 100, 0.05
		);
		//printf("Mean reward: %.2f | final distance: %.2f | inital distance: %.2f || ", mean_reward, (abs(target_x - x) + abs(target_y - y)), abs(target_x) + abs(target_y));
		int reward_pos = (int)(mean_reward * 10 + 10);
		printf("       ");
		for (size_t j = 0; j < 21; j++) reward_pos == j ? printf("#") : printf("-");
		
		data_t total_mean_r = 0;
		for (size_t j = 0; j < i + 1; j++) total_mean_r += epochs_mean_reward[j];

		printf("  Total mean reward: %.5f | %i\n", total_mean_r / (i + 1), i);

		delete[] Y;
	}
	delete n;
	//n.deallocate();
}
