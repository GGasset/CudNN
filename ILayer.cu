#include "ILayer.h"

size_t ILayer::get_neuron_count()
{
	return neuron_count;
}

void ILayer::set_neuron_count(size_t neuron_count)
{
	this->neuron_count = neuron_count;
	connections->neuron_count = neuron_count;
}

void ILayer::initialize_fields(size_t connection_count, size_t neuron_count)
{
	layer_specific_initialize_fields(connection_count, neuron_count);
	hipDeviceSynchronize();
}

void ILayer::layer_specific_initialize_fields(size_t connection_count, size_t neuron_count)
{
}

void ILayer::ILayerClone(ILayer* base_layer)
{
	IConnections* cloned_connections = connections->connections_specific_clone();
	connections->IConnections_clone(cloned_connections);
	base_layer->connections = cloned_connections;

	base_layer->set_neuron_count(get_neuron_count());

	base_layer->execution_values_per_neuron = execution_values_per_neuron;
	
	base_layer->layer_derivative_count = layer_derivative_count;
	base_layer->derivatives_per_neuron = derivatives_per_neuron;

	base_layer->layer_gradient_count = layer_gradient_count;
	
	hipMalloc(&base_layer->neuron_gradients_starts, sizeof(size_t) * get_neuron_count());
	if (connection_associated_gradient_counts)
		hipMalloc(&base_layer->connection_associated_gradient_counts, sizeof(size_t) * get_neuron_count());
	hipDeviceSynchronize();

	hipMemcpy(base_layer->neuron_gradients_starts, neuron_gradients_starts, sizeof(size_t) * get_neuron_count(), hipMemcpyDeviceToDevice);
	if (connection_associated_gradient_counts)
		hipMemcpy(base_layer->connection_associated_gradient_counts, connection_associated_gradient_counts, sizeof(size_t) * neuron_count, hipMemcpyDeviceToDevice);
}

void ILayer::save(FILE* file)
{
	fwrite(&neuron_count, sizeof(size_t), 1, file);
	fwrite(&execution_values_per_neuron, sizeof(size_t), 1, file);
	fwrite(&layer_derivative_count, sizeof(size_t), 1, file);
	fwrite(&derivatives_per_neuron, sizeof(size_t), 1, file);
	fwrite(&layer_gradient_count, sizeof(size_t), 1, file);
	
	size_t *host_neuron_gradients_starts, *host_connection_gradient_counts;
	host_neuron_gradients_starts = new size_t[neuron_count];
	host_connection_gradient_counts = new size_t[neuron_count];

	hipMemcpy(host_neuron_gradients_starts, neuron_gradients_starts, sizeof(size_t) * neuron_count, hipMemcpyDeviceToHost);
	hipMemcpy(host_connection_gradient_counts, connection_associated_gradient_counts, sizeof(size_t) * neuron_count, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	fwrite(host_neuron_gradients_starts, sizeof(size_t), neuron_count, file);
	fwrite(host_connection_gradient_counts, sizeof(size_t), neuron_count, file);
	delete[] host_neuron_gradients_starts;
	delete[] host_connection_gradient_counts;

	specific_save(file);
}

void ILayer::ILayer_load(FILE* file)
{
	fread(&neuron_count, sizeof(size_t), 1, file);
	fread(&execution_values_per_neuron, sizeof(size_t), 1, file);
	fread(&layer_derivative_count, sizeof(size_t), 1, file);
	fread(&derivatives_per_neuron, sizeof(size_t), 1, file);
	fread(&layer_gradient_count, sizeof(size_t), 1, file);

	size_t* host_neuron_gradients_starts = new size_t[neuron_count];
	size_t* host_connection_associated_gradient_counts = new size_t[neuron_count];

	fread(host_neuron_gradients_starts, sizeof(size_t), neuron_count, file);
	fread(host_connection_associated_gradient_counts, sizeof(size_t), neuron_count, file);

	hipMalloc(&neuron_gradients_starts, sizeof(size_t) * neuron_count);
	hipMalloc(&connection_associated_gradient_counts, sizeof(size_t) * neuron_count);
	hipDeviceSynchronize();

	hipMemcpy(neuron_gradients_starts, host_neuron_gradients_starts, sizeof(size_t) * neuron_count, hipMemcpyHostToDevice);
	hipMemcpy(connection_associated_gradient_counts, host_connection_associated_gradient_counts, sizeof(size_t) * neuron_count, hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	delete[] host_neuron_gradients_starts;
	delete[] host_connection_associated_gradient_counts;
}

void ILayer::deallocate()
{
	connections->deallocate();
	layer_specific_deallocate();
	hipDeviceSynchronize();
	delete connections;
}

void ILayer::layer_specific_deallocate()
{

}

void ILayer::mutate_fields(evolution_metadata evolution_values)
{
}

void ILayer::adjust_to_added_neuron(size_t added_neuron_i, float connection_probability)
{
	auto added_connections_neuron_i = std::vector<size_t>();
	connections->adjust_to_added_neuron(added_neuron_i, connection_probability, &added_connections_neuron_i);
	for (size_t i = 0; i < added_connections_neuron_i.size(); i++)
	{
		layer_gradient_count++;
		size_t added_connection_neuron_i = added_connections_neuron_i[i];
		size_t remaining_neuron_count = neuron_count - added_connection_neuron_i - 1;
		if (remaining_neuron_count)
		{
			if (connection_associated_gradient_counts)
				add_to_array kernel(1, 1) (
					connection_associated_gradient_counts + added_connection_neuron_i, 1, 1
				);
			if (neuron_gradients_starts)
				add_to_array kernel(remaining_neuron_count / 32 + (remaining_neuron_count % 32 > 0), 32) (
					neuron_gradients_starts + added_connection_neuron_i + 1, remaining_neuron_count, 1
				);
		}
	}
}

void ILayer::adjust_to_removed_neuron(size_t neuron_i)
{
	auto removed_connections_neuron_i = std::vector<size_t>();
	connections->adjust_to_removed_neuron(neuron_i, &removed_connections_neuron_i);
	for (size_t i = 0; i < removed_connections_neuron_i.size(); i++)
	{
		layer_gradient_count--;
		size_t removed_connection_neuron_i = removed_connections_neuron_i[i];
		size_t remaining_neuron_count = neuron_count - removed_connection_neuron_i - 1;
		if (remaining_neuron_count)
		{
			if (connection_associated_gradient_counts)
				add_to_array kernel(1, 1) (
					connection_associated_gradient_counts + removed_connection_neuron_i, 1, -1
				);
			if (neuron_gradients_starts)
				add_to_array kernel(remaining_neuron_count / 32 + (remaining_neuron_count % 32 > 0), 32) (
					neuron_gradients_starts + removed_connection_neuron_i + 1, remaining_neuron_count, -1
				);
		}
	}
}

void ILayer::delete_memory()
{
}
