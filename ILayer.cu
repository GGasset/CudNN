#ifndef ILAYER_DEFINITIONS
#define ILAYER_DEFINITIONS

#include "ILayer.h"

size_t ILayer::get_neuron_count()
{
	return neuron_count;
}

void ILayer::set_neuron_count(size_t neuron_count)
{
	this->neuron_count = neuron_count;
	connections->neuron_count = neuron_count;
}

void ILayer::initialize_fields(size_t connection_count, size_t neuron_count)
{
	layer_specific_initialize_fields(connection_count, neuron_count);
	hipDeviceSynchronize();
}

void ILayer::layer_specific_initialize_fields(size_t connection_count, size_t neuron_count)
{
}

void ILayer::ILayerClone(ILayer* base_layer)
{
	IConnections* cloned_connections = connections->connections_specific_clone();
	connections->IConnections_clone(cloned_connections);
	base_layer->connections = cloned_connections;

	base_layer->set_neuron_count(get_neuron_count());

	base_layer->execution_values_per_neuron = execution_values_per_neuron;
	
	base_layer->layer_derivative_count = layer_derivative_count;
	base_layer->derivatives_per_neuron = derivatives_per_neuron;

	base_layer->layer_gradient_count = layer_gradient_count;
	
	hipMalloc(&base_layer->neuron_gradients_starts, sizeof(size_t) * get_neuron_count());
	if (connection_associated_gradient_counts)
		hipMalloc(&base_layer->connection_associated_gradient_counts, sizeof(size_t) * get_neuron_count());
	hipDeviceSynchronize();

	hipMemcpy(base_layer->neuron_gradients_starts, neuron_gradients_starts, sizeof(size_t) * get_neuron_count(), hipMemcpyDeviceToDevice);
	if (connection_associated_gradient_counts)
		hipMemcpy(base_layer->connection_associated_gradient_counts, connection_associated_gradient_counts, sizeof(size_t) * neuron_count, hipMemcpyDeviceToDevice);
}

void ILayer::save(FILE* file)
{
	fwrite(&neuron_count, sizeof(size_t), 1, file);
	fwrite(&execution_values_per_neuron, sizeof(size_t), 1, file);
	fwrite(&layer_derivative_count, sizeof(size_t), 1, file);
	fwrite(&derivatives_per_neuron, sizeof(size_t), 1, file);
	fwrite(&layer_gradient_count, sizeof(size_t), 1, file);
	
	size_t *host_neuron_gradients_starts, *host_connection_gradient_counts;
	host_neuron_gradients_starts = new size_t[neuron_count];
	host_connection_gradient_counts = new size_t[neuron_count];

	hipMemcpy(host_neuron_gradients_starts, neuron_gradients_starts, sizeof(size_t) * neuron_count, hipMemcpyDeviceToHost);
	hipMemcpy(host_connection_gradient_counts, connection_associated_gradients_counts, sizeof(size_t) * neuron_count, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	fwrite(host_neuron_gradients_starts, sizeof(size_t), neuron_count, file);
	fwrite(host_connection_gradient_counts, sizeof(size_t), neuron_count, file);
	delete[] host_neuron_gradients_starts;
	delete[] host_connection_gradient_counts;

	specific_save(file);
}

void ILayer::deallocate()
{
	connections->deallocate();
	layer_specific_deallocate();
	hipDeviceSynchronize();
	delete connections;
}

void ILayer::layer_specific_deallocate()
{

}

void ILayer::mutate_fields(evolution_metadata evolution_values)
{
}

void ILayer::delete_memory()
{
}

#endif
