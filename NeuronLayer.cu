#ifndef NEURONLAYER_DEFINITIONS
#define NEURONLAYER_DEFINITIONS

#include "NeuronLayer.h"
#include <stdio.h>

NeuronLayer::NeuronLayer(IConnections* connections, size_t neuron_count, ActivationFunctions activation)
{
	layer_type = NeuronTypes::Neuron;

	this->connections = connections;
	set_neuron_count(neuron_count);
	this->activation = activation;
	execution_values_per_neuron = 1;
	layer_gradient_count = connections->connection_count + neuron_count;

	initialize_fields(connections->connection_count, neuron_count, false);
}

NeuronLayer::NeuronLayer()
{
	layer_type = NeuronTypes::Neuron;
}

void NeuronLayer::layer_specific_deallocate()
{
	hipFree(neuron_gradients_starts);
	if (connection_associated_gradient_counts)
		hipFree(connection_associated_gradient_counts);
}

ILayer* NeuronLayer::layer_specific_clone()
{
	NeuronLayer* layer = new NeuronLayer();
	layer->activation = activation;
	return layer;
}

void NeuronLayer::specific_save(FILE* file)
{
	size_t activation_function = (size_t)activation;
	fwrite(&activation_function, sizeof(size_t), 1, file);
}

void NeuronLayer::load(FILE* file)
{
	ILayer_load(file);
	
	size_t activation_function = 0;
	fread(&activation_function, sizeof(size_t), 1, file);
	activation = (ActivationFunctions)activation_function;
}

void NeuronLayer::execute(
	data_t* activations, size_t activations_start,
	data_t* execution_values, size_t execution_values_start
)
{
	connections->linear_function(activations_start, activations,
		execution_values, execution_values_start, execution_values_layer_start, execution_values_per_neuron
	);
	switch (activation)
	{
	case ActivationFunctions::sigmoid:
		sigmoid_activation kernel(neuron_count / 32 + (neuron_count % 32 > 0), 32) (
			activations, activations_start, layer_activations_start, true,
			execution_values, execution_values_start, execution_values_layer_start, execution_values_per_neuron, 0, 0, 0,
			neuron_count
		);
		break;
	case ActivationFunctions::_tanh:
		tanh_activation kernel(neuron_count / 32 + (neuron_count % 32 > 0), 32) (
			activations, activations_start, layer_activations_start, true,
			execution_values, execution_values_start, execution_values_layer_start, execution_values_per_neuron, 0, 0, 0,
			neuron_count
		);
		break;
	default:
		break;
	}
	hipDeviceSynchronize();
}

void NeuronLayer::calculate_gradients(
	data_t* activations, size_t activations_start,
	data_t* execution_values, size_t execution_values_start,
	data_t* derivatives, size_t derivatives_start,
	data_t* gradients, size_t next_gradients_start, size_t gradients_start,
	data_t* costs, size_t costs_start
)
{
	neuron_gradient_calculation kernel(neuron_count / 32 + (neuron_count % 32 > 0), 32) (
		execution_values, execution_values_start, execution_values_layer_start,
		gradients, gradients_start, layer_gradients_start, neuron_gradients_starts,
		costs, costs_start, layer_activations_start,
		activation,
		neuron_count
	);
	hipDeviceSynchronize();
	connections->calculate_gradients(
		activations, activations_start, gradients, gradients_start, layer_gradients_start, neuron_gradients_starts,
		costs, costs_start
	);
	hipDeviceSynchronize();
}

void NeuronLayer::subtract_gradients(data_t* gradients, size_t gradients_start, data_t learning_rate, short* dropout, data_t gradient_clip)
{
	connections->subtract_gradients(
		gradients, gradients_start, layer_gradients_start, neuron_gradients_starts,
		learning_rate, dropout, gradient_clip
	);
}

#endif
