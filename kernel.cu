#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

#include "NN_constructor.h"

/*
static int increment_i(size_t size, int to_increment, size_t i)
{
	i += to_increment;+
	i -= (i - i % size) * (i >= size);
	i = (i % size) * (i < 0) + i * (i >= 0);
	return i;
}*/

template<typename t>
t abs(t a)
{
	return a * (-1 + 2 * (a > 0));
}

int main()
{
	srand(101);
	hipSetDevice(0);

	const size_t input_length = 2;
	const size_t output_length = 2;
	/*const size_t t_count = 5;
	data_t X[input_length * t_count]{};
	data_t Y_hat[output_length * t_count]{};

	for (size_t t = 0; t < t_count; t++)
	{
		for (size_t i = 0; i < input_length; i++)
		{
			X[t * input_length + i] = (.2);//  / t_count * ((t + 1)) + .2 / t_count;
		}
		 for (size_t i = 0; i < output_length; i++)
		{
			Y_hat[t * output_length + i] = .05;//(.2) / t_count * (t + 1) + .2 / t_count + (.2) / t_count * (i + 1) / output_length;
		}
	}*/


	
	NN* n = NN_constructor()
		.append_layer(ConnectionTypes::Dense, NeuronTypes::Neuron, 30, ActivationFunctions::sigmoid)
		.append_layer(ConnectionTypes::Dense, NeuronTypes::Neuron, 15, ActivationFunctions::sigmoid)
		.append_layer(ConnectionTypes::Dense, NeuronTypes::Neuron, 7, ActivationFunctions::sigmoid)
		.append_layer(ConnectionTypes::Dense, NeuronTypes::Neuron, 5, ActivationFunctions::sigmoid)
		.append_layer(ConnectionTypes::Dense, NeuronTypes::Neuron, output_length, ActivationFunctions::sigmoid)
		.construct(input_length);
	/*
	NN* n = NN_constructor()
		.append_layer(ConnectionTypes::Dense, NeuronTypes::Neuron, output_length, ActivationFunctions::sigmoid)
		.construct(input_length);
	*/
	n->stateful = true;
	
	data_t total_mean_r = 0;
	data_t max_total_mean_r = -100000;
	size_t total_mean_r_count = 0;

	const size_t epochs = 100000;
	const size_t max_steps = 40;
	for (size_t i = 0; i < epochs; i++)
	{
		// RL demonstration
		data_t x = 0;
		data_t y = 0;

		data_t target_x = (rand() % 50 + 50);
		data_t target_y = (rand() % 50 + 50);
		target_x *= 1 - 2 * (rand() % 2);
		target_y *= 1 - 2 * (rand() % 2);

		data_t X[2];
		data_t *Y = 0;
		data_t* execution_values = 0;
		data_t* activations = 0;
		
		data_t rewards[max_steps];
		data_t supervised_outputs[max_steps * 2];
		
		bool success = false;
		size_t actual_steps = 0;
		data_t max_reward = 0;
		data_t mean_reward = 0;
		data_t mean_output[output_length] {0, 0};
		for (size_t step_i = 0; step_i < max_steps; step_i++)
		{
			
			target_x = (rand() % 50 + 50);
			target_y = (rand() % 50 + 50);
			target_x *= 1 - 2 * (rand() % 2);
			target_y *= 1 - 2 * (rand() % 2);

			actual_steps++;
			rewards[step_i] = 0;

			data_t target_direction_x = target_x - x;
			data_t target_direction_y = target_y - y;
			data_t target_distance = abs(target_direction_x) + abs(target_direction_y);


			X[0] = (target_direction_x > 0 ? .5 : -.5) * 1;
			X[1] = (target_direction_y > 0 ? .5 : -.5) * 1;

			supervised_outputs[step_i * 2] = target_direction_x > 0 ? .75 : .25;
			supervised_outputs[step_i * 2 + 1] = target_direction_y > 0 ? .75 : .25;
			
			n->training_execute(
				1, X, &Y, true,
				&execution_values, &activations,
				step_i
			);

			//x += (Y[0] - .5) * 2 * 3;
			//y += (Y[1] - .5) * 2 * 3;
			x += (Y[0] > .5) ? 1 : -1;
			y += (Y[1] > .5) ? 1 : -1;

			mean_output[0] += Y[0];
			mean_output[1] += Y[1];


			data_t new_target_direction_x = target_x - x;
			data_t new_target_direction_y = target_y - y;
			data_t new_target_distance = abs(new_target_direction_x) + abs(new_target_direction_y);

			//rewards[step_i]--;
			rewards[step_i] += 
				((abs(target_direction_x) > abs(new_target_direction_x))
				+ (abs(target_direction_y) > abs(new_target_direction_y)))
				* .5;
			
			rewards[step_i] -= rewards[step_i] == 0;

			max_reward += (abs(rewards[step_i]) - max_reward) * (abs(rewards[step_i]) > max_reward);
			mean_reward += rewards[step_i];

			if (success = abs(new_target_distance) < 3 && false)
			{
				printf("Hit!   ");
				break;
			}
		}
		
		mean_output[0] /= actual_steps;
		mean_output[1] /= actual_steps;
		
		mean_reward /= actual_steps;

		data_t value_functions[max_steps];
		const data_t gamma = .995;
		for (size_t j = 0; j < actual_steps; j++)
		{
			value_functions[j] = 0;
			data_t discount_factor = 1;
			
			for (size_t k = j; k < actual_steps; k++, discount_factor *= gamma)
			{
				value_functions[j] += rewards[k] * discount_factor;
			}
		}

		// Denoted as greek lowercase delta
		data_t deltas[max_steps];
		for (size_t j = 0; j < actual_steps; j++)
		{
			data_t discount_factor = gamma;
			deltas[j] = -value_functions[j] + rewards[j] + discount_factor * (j == actual_steps - 1 ? 0 : value_functions[j + 1]);
		}

		const data_t lambda = .98;
		data_t gamma_lambda = gamma * lambda;
		data_t advantages[max_steps];
		for (size_t j = 0; j < actual_steps; j++)
		{
			advantages[j] = 0;
			data_t GAE_discount = 1;
			for (size_t k = j; k < actual_steps && (/*Handle lambda=0 for performace*/1); k++, GAE_discount *= gamma_lambda)
			{
				advantages[j] += GAE_discount * deltas[k];
			}
		}
		

		/*n->train(actual_steps,
			execution_values, activations, 
			advantages, true, actual_steps,
			CostFunctions::log_likelyhood, .5, 100, 0.2
		);*/

		n->train(actual_steps,
			execution_values, activations, 
			supervised_outputs, true, actual_steps * output_length,
			CostFunctions::MSE, .001, 100, .2
		);



		//printf("Mean reward: %.2f | final distance: %.2f | inital distance: %.2f || ", mean_reward, (abs(target_x - x) + abs(target_y - y)), abs(target_x) + abs(target_y));
		int reward_pos = (int)((1 + mean_reward) * 10);
		for (size_t j = 0; j <= 20; j++) reward_pos == j ? printf("#") : printf("-");
		
		total_mean_r += mean_reward;
		total_mean_r_count++;

		printf("  Current mean reward: %.7f | Mean Reward: %.2f | target_x: %.0f, target_y: %.0f | %i | %.3f", mean_reward, total_mean_r / total_mean_r_count, target_x, target_y, i, max_total_mean_r);
		printf(" | Mean x output: %.2f | Mean y output %.2f\n", mean_output[0], mean_output[1]);

		if (i % 100 == 0)
		{
			max_total_mean_r += ((total_mean_r / total_mean_r_count) - max_total_mean_r) * (total_mean_r > max_total_mean_r);
			total_mean_r = total_mean_r_count = 0;
		}
		delete[] Y;
	}
	delete n;
	hipDeviceReset();
	//n.deallocate();
}
