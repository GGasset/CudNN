#include "NeatConnections.h"
#include "cuda_functionality.cuh"
#include <cstddef>

NeatConnections::NeatConnections(size_t previous_layer_start, size_t previous_layer_length, size_t neuron_count)
{
	contains_irregular_connections = true;
	this->neuron_count = neuron_count;
	this->connection_count = neuron_count * previous_layer_length;
	hipMalloc(&weights, sizeof(field_t) * connection_count);
	hipMalloc(&biases, sizeof(field_t) * neuron_count);
	hipMalloc(&connection_points, sizeof(size_t) * connection_count);
	hipMalloc(&connection_neuron_i, sizeof(size_t) * connection_count);
	hipDeviceSynchronize();

	generate_random_values(&weights, neuron_count * previous_layer_length, 0, previous_layer_length);
	hipMemset(biases, 0, sizeof(field_t) * neuron_count);
	//generate_random_values(&biases, neuron_count, 0, neuron_count);
	
	size_t* host_connection_points = new size_t[connection_count];
	size_t* host_connection_neuron_i = new size_t[connection_count];
	//connection_counts = new size_t[neuron_count];
	for (size_t i = 0; i < neuron_count; i++)
	{
		for (size_t j = 0; j < previous_layer_length; j++)
		{
			host_connection_points[i * previous_layer_length + j] = previous_layer_start + j;
			host_connection_neuron_i[i * previous_layer_length + j] = i;
		}
		//connection_counts[i] = previous_layer_length;
	}
	hipMemcpy(connection_points, host_connection_points, sizeof(size_t) * connection_count, hipMemcpyHostToDevice);
	hipMemcpy(connection_neuron_i, host_connection_neuron_i, sizeof(size_t) * connection_count, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
  delete[] host_connection_neuron_i;
	delete[] host_connection_points;
}

NeatConnections::NeatConnections()
{

}

void NeatConnections::linear_function(
	size_t activations_start, data_t* activations, 
	data_t* execution_values, size_t execution_values_start, size_t execution_values_layer_start, size_t layer_execution_values_per_neuron
)
{
	cud_NEAT_linear_function kernel(connection_count / 32 + (connection_count % 32 > 0), 32) (
		connection_count, weights, connection_points, connection_neuron_i,
		activations_start, activations,
		execution_values_start, execution_values_layer_start, layer_execution_values_per_neuron, execution_values
	);
	cud_add_biases kernel(dim3(neuron_count / 32 + (neuron_count % 32 > 0), 1, 1), 32) (
		neuron_count, biases, 
		execution_values_start, execution_values_layer_start, layer_execution_values_per_neuron, execution_values
	);
	hipDeviceSynchronize();
}

void NeatConnections::calculate_derivative(
	size_t activations_start, data_t* activations, 
	size_t derivatives_start, size_t derivatives_layer_start, size_t derivatives_per_neuron, data_t* derivatives
)
{
	cud_NEAT_linear_function_derivative kernel(connection_count / 32 + (connection_count % 32 > 0), 32) (
		activations_start, activations,
		derivatives_start, derivatives_layer_start, derivatives_per_neuron, derivatives,
		connection_count, weights, connection_points, connection_neuron_i
	);

	cud_add_bias_derivative kernel(neuron_count / 32 + (neuron_count % 32 > 0), 32) (
		neuron_count, derivatives_start, derivatives_layer_start, derivatives_per_neuron, derivatives
	);
	hipDeviceSynchronize();
}

void NeatConnections::calculate_gradients(
	data_t* activations, size_t activations_start, 
	data_t* gradients, size_t gradients_start, size_t layer_gradients_start, size_t* neuron_gradients_starts, 
	data_t* costs, size_t costs_start
)
{
	cud_NEAT_gradient_calculation kernel(connection_count / 32 + (connection_count % 32 > 0), 32) (
		activations, activations_start,
		gradients, gradients_start, layer_gradients_start, neuron_gradients_starts,
		costs, costs_start,
		connection_count, weights, connection_points, connection_neuron_i
	);
	/*size_t connections_start = 0;
	for (size_t i = 0; i < neuron_count; i++)
	{
		size_t connection_count = connection_counts[i];
		cud_NEAT_gradient_calculation kernel(connection_count / 32 + (connection_count % 32 > 0), 32) (
			activations, activations_start,
			gradients, gradients_start, layer_gradients_start, neuron_gradients_starts,
			costs, costs_start,
			i, connection_count, weights, connection_points, connections_start
		);

		connections_start += connection_count;
	}*/
	hipDeviceSynchronize();
}

void NeatConnections::subtract_gradients(
	data_t* gradients, size_t gradients_start, size_t layer_gradients_start, size_t* neuron_gradients_starts, 
	data_t learning_rate, short* dropout, data_t gradient_clip
)
{
	/*size_t connections_start = 0;
	for (size_t i = 0; i < neuron_count; i++)
	{
		size_t connection_count = connection_counts[i];
		cud_NEAT_gradient_subtraction kernel(connection_count / 32 + (connection_count % 32 + 1), 32) (
			gradients, gradients_start, layer_gradients_start, neuron_gradients_starts,
			i, connection_count, weights, connections_start,
			learning_rate, dropout, gradient_clip
		);
		connections_start += connection_count;
	}*/
	cud_NEAT_gradient_subtraction kernel(connection_count / 32 + (connection_count % 32 > 0), 32) (
		gradients, gradients_start, layer_gradients_start, neuron_gradients_starts,
		connection_neuron_i, connection_count,
		weights,
		learning_rate, dropout, gradient_clip
	);
	bias_gradient_subtraction kernel(neuron_count / 32 + (neuron_count % 32 > 0), 32) (
		gradients, gradients_start, layer_gradients_start, neuron_gradients_starts,
		biases, neuron_count, learning_rate, dropout, gradient_clip
	);
	hipDeviceSynchronize();
}

size_t NeatConnections::get_connection_count_at(size_t neuron_i)
{
  unsigned int* device_count = 0;
  hipMalloc(&device_count, sizeof(unsigned int));
  hipDeviceSynchronize();
  hipMemset(device_count, 0, sizeof(unsigned int));
	count_value kernel(connection_count / 32 + (connection_count % 32 > 0), 32) (neuron_i, connection_neuron_i, connection_count, device_count);
  hipDeviceSynchronize();
  unsigned int count = 0;
  hipMemcpy(&count, device_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  hipFree(device_count);
  return (size_t)count;
  //return connection_counts[neuron_i];
}

void NeatConnections::add_neuron(size_t previous_layer_length, size_t previous_layer_activations_start, float previous_layer_connection_probability, size_t min_connections)
{
	size_t added_connection_count = 0;
	size_t* device_tmp_connections = 0;
	field_t* device_tmp_biases = 0;
	field_t* device_tmp_weights = 0;
	size_t* tmp_connections_neuron_i = 0;
  auto tmp_connections = std::vector<size_t>();
	auto sampling_vector = std::vector<size_t>();
	for (size_t i = 0; i < previous_layer_length; i++)
	{
		if (rand() % 100000 / 100000.0 < previous_layer_connection_probability)
		{
			added_connection_count++;
			tmp_connections.push_back(i + previous_layer_activations_start);
			continue;
		}
		sampling_vector.push_back(i + previous_layer_activations_start);
	}
	while (tmp_connections.size() < min_connections && sampling_vector.size())
	{
		size_t i = rand() % sampling_vector.size();
		tmp_connections.push_back(sampling_vector.at(i));
		sampling_vector.erase(sampling_vector.begin() + i);
		added_connection_count++;
	}
	
	hipMalloc(&device_tmp_connections, sizeof(size_t) * (connection_count + added_connection_count));
	hipMalloc(&device_tmp_weights, sizeof(field_t) * (connection_count + added_connection_count));
	hipMalloc(&device_tmp_biases, sizeof(field_t) * (neuron_count + 1));
  hipMalloc(&tmp_connections_neuron_i , sizeof(size_t) * (connection_count + added_connection_count));
	hipDeviceSynchronize();

	hipMemcpy(device_tmp_weights, weights, sizeof(field_t) * connection_count, hipMemcpyDeviceToDevice);
  
  hipMemcpy(tmp_connections_neuron_i, connection_neuron_i, sizeof(size_t) * connection_count, hipMemcpyDeviceToDevice);
  hipMemset(tmp_connections_neuron_i + connection_count, 0, sizeof(size_t) * added_connection_count);

	hipMemcpy(device_tmp_biases, biases, sizeof(field_t) * neuron_count, hipMemcpyDeviceToDevice);


	hipMemcpy(device_tmp_connections, connection_points, sizeof(size_t) * connection_count, hipMemcpyDeviceToDevice);
	hipMemcpy(device_tmp_connections + connection_count, tmp_connections.data(), sizeof(size_t) * added_connection_count, hipMemcpyHostToDevice);
	hipDeviceSynchronize();

  add_to_array kernel(added_connection_count / 32 + (added_connection_count % 32 > 0), 32) (tmp_connections_neuron_i + connection_count, added_connection_count, neuron_count);
	generate_random_values(&device_tmp_biases, 1, neuron_count);
	generate_random_values(&device_tmp_weights, added_connection_count, connection_count);

	hipFree(weights);
	hipFree(biases);
	hipFree(connection_points);
	hipDeviceSynchronize();
	
  
	/*size_t* tmp_connection_counts = new size_t[neuron_count + 1];
	hipMemcpy(tmp_connection_counts, connection_counts, sizeof(size_t) * neuron_count, hipMemcpyHostToHost);
	hipDeviceSynchronize();
	tmp_connection_counts[neuron_count] = added_connection_count;
	delete[] connection_counts;*/

	//connection_counts = tmp_connection_counts;
	connection_neuron_i = tmp_connections_neuron_i;
  weights = device_tmp_weights;
	biases = device_tmp_biases;
	connection_points = device_tmp_connections;
	connection_count += added_connection_count;
}

void NeatConnections::adjust_to_added_neuron(size_t added_neuron_i, float connection_probability, std::vector<size_t>* added_connections_neuron_i)
{
	// Transform data to a vector
	size_t *host_connection_points = new size_t[connection_count];
  size_t *host_connection_neuron_i = new size_t[connection_count];
	field_t* host_weights = new field_t[connection_count];
	
	hipMemcpy(host_connection_points, connection_points, sizeof(size_t) * connection_count, hipMemcpyDeviceToHost);
	hipMemcpy(host_weights, weights, sizeof(field_t) * connection_count, hipMemcpyDeviceToHost);
	hipMemcpy(host_connection_neuron_i, connection_neuron_i, sizeof(size_t) * connection_count, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	auto vector_connection_points = std::vector<size_t>();
	auto vector_weights = std::vector<field_t>();
  std::vector<size_t> vector_connection_neuron_i;
	for (size_t i = 0; i < connection_count; i++)
	{
		// Adjust connections for index change while transforming points to a vector
		vector_connection_points.push_back(host_connection_points[i] + (host_connection_points[i] >= added_neuron_i));
		vector_weights.push_back(host_weights[i]);
    vector_connection_neuron_i.push_back(host_connection_neuron_i[i]);
	}

	// Add connections
	size_t tmp_connection_count = 0;
	for (size_t i = 0; i < neuron_count; i++)
	{
		uint8_t is_connection_added = rand() % 100000 / 100000.0 < connection_probability;
		size_t old_neuron_connection_count = get_connection_count_at(i);
		size_t new_neuron_connection_count = old_neuron_connection_count + is_connection_added;
		//connection_counts[i] = new_neuron_connection_count;
		
		if (is_connection_added)
		{
			added_connections_neuron_i->push_back(i);
			vector_connection_points.insert(vector_connection_points.begin() + tmp_connection_count + old_neuron_connection_count, added_neuron_i);
			vector_weights.insert(vector_weights.begin() + tmp_connection_count + old_neuron_connection_count, rand() % 100000 / 100000.0);
      vector_connection_neuron_i.insert(vector_connection_neuron_i.begin() + tmp_connection_count + old_neuron_connection_count, i);
		}

		connection_count += is_connection_added;
		tmp_connection_count += new_neuron_connection_count;
	}

	// Copy data to device, free old arrays and free host arrays
  hipFree(connection_neuron_i);
	hipFree(connection_points);
	hipFree(weights);
	delete[] host_connection_points;
	delete[] host_weights;
	hipDeviceSynchronize();

  hipMalloc(&connection_neuron_i, sizeof(size_t) * connection_count);
	hipMalloc(&weights, sizeof(field_t) * connection_count);
	hipMalloc(&connection_points, sizeof(size_t) * connection_count);
	hipDeviceSynchronize();

  hipMemcpy(connection_neuron_i, vector_connection_neuron_i.data(), sizeof(size_t) * connection_count, hipMemcpyHostToDevice);
	hipMemcpy(weights, vector_weights.data(), sizeof(field_t) * connection_count, hipMemcpyHostToDevice);
	hipMemcpy(connection_points, vector_connection_points.data(), sizeof(size_t) * connection_count, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
}

void NeatConnections::remove_neuron(size_t neuron_i)
{
  size_t* tmp_connections_neuron_i = 0;
	size_t* tmp_connection_points = 0;
	field_t* tmp_weights = 0;
	field_t* tmp_biases = 0;

	size_t connection_count_until_deletion = 0;
	for (size_t i = 0; i < neuron_i; i++)
		connection_count_until_deletion += get_connection_count_at(i);

	size_t connection_count_after_deletion = 0;
	for (size_t i = neuron_i + 1; i < neuron_count; i++)
		connection_count_after_deletion += get_connection_count_at(i);

	size_t to_delete_connection_count = get_connection_count_at(neuron_i);
	
  hipMalloc(&tmp_connections_neuron_i, sizeof(size_t) * (connection_count - to_delete_connection_count));
	hipMalloc(&tmp_connection_points, sizeof(size_t) * (connection_count - to_delete_connection_count));
	hipMalloc(&tmp_weights, sizeof(field_t) * (connection_count - to_delete_connection_count));
	hipMalloc(&tmp_biases, sizeof(field_t) * (neuron_count - 1));
	hipDeviceSynchronize();

  hipMemcpy(tmp_connections_neuron_i, connection_neuron_i, sizeof(size_t) * (connection_count_until_deletion), hipMemcpyDeviceToDevice);
  hipMemcpy(tmp_connections_neuron_i + connection_count_until_deletion, connection_neuron_i + connection_count_until_deletion + to_delete_connection_count, sizeof(size_t) * connection_count_after_deletion, hipMemcpyDeviceToDevice);
  
	hipMemcpy(tmp_connection_points, connection_points, sizeof(size_t) * connection_count_until_deletion, hipMemcpyDeviceToDevice);
	hipMemcpy(tmp_connection_points + connection_count_until_deletion, connection_points + connection_count_until_deletion + to_delete_connection_count, sizeof(size_t) * connection_count_after_deletion, hipMemcpyDeviceToDevice);

	hipMemcpy(tmp_weights, weights, sizeof(field_t) * connection_count_until_deletion, hipMemcpyDeviceToDevice);
	hipMemcpy(tmp_weights + connection_count_until_deletion, weights + connection_count_until_deletion + to_delete_connection_count, sizeof(field_t) * connection_count_after_deletion, hipMemcpyDeviceToDevice);

	hipMemcpy(tmp_biases, biases, sizeof(field_t) * neuron_i, hipMemcpyDeviceToDevice);
	hipMemcpy(tmp_biases + neuron_i, biases + neuron_i + 1, sizeof(field_t) * (neuron_count - neuron_i - 1), hipMemcpyDeviceToDevice);
	hipDeviceSynchronize();

	hipFree(connection_points);
	hipFree(weights);
	hipFree(biases);
  hipFree(connection_neuron_i);
	hipDeviceSynchronize();

  connection_neuron_i = tmp_connections_neuron_i;
	connection_points = tmp_connection_points;
	weights = tmp_weights;
	biases = tmp_biases;
	connection_count -= to_delete_connection_count;
}

void NeatConnections::adjust_to_removed_neuron(size_t neuron_i, std::vector<size_t>* removed_connections_neuron_i)
{
	size_t* host_connection_points = new size_t[connection_count];
  size_t* host_connection_neuron_i = new size_t[connection_count];
	field_t* host_weights = new field_t[connection_count];

	hipMemcpy(host_connection_points, connection_points, sizeof(size_t) * connection_count, hipMemcpyDeviceToHost);
	hipMemcpy(host_weights, weights, sizeof(field_t) * connection_count, hipMemcpyDeviceToHost);
  hipMemcpy(host_connection_neuron_i, connection_neuron_i, sizeof(size_t) * connection_count, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	auto connection_points_vector = std::vector<size_t>();
	auto vector_weights = std::vector<field_t>();
  std::vector<size_t> vector_connection_neuron_i;
	for (size_t i = 0; i < connection_count; i++)
	{
		// Adjust connections for index change while transforming points to a vector
		connection_points_vector.push_back(host_connection_points[i]);
		vector_weights.push_back(host_weights[i]);
    vector_connection_neuron_i.push_back(host_connection_neuron_i[i]);
	}

  delete[] host_connection_points;
  delete[] host_connection_neuron_i;
  delete[] host_weights;

	size_t found_i = 0;
	while (true)
	{
		// Search for connections pointing to neuron_i, break if not found
		uint8_t found = false;
		for (size_t i = found_i; i < connection_count && !found; i++)
		{
			found = connection_points_vector[i] == neuron_i;
			found_i = i;
		}
		if (!found)
			break;

		// Get the neuron containing the connection
		size_t connection_neuron_i = vector_connection_neuron_i[found_i];

		// Update info
		removed_connections_neuron_i->push_back(connection_neuron_i);
		vector_weights.erase(vector_weights.begin() + found_i);
		connection_points_vector.erase(connection_points_vector.begin() + found_i);
    vector_connection_neuron_i.erase(vector_connection_neuron_i.begin() + found_i);
		connection_count--;
	}
	for (size_t i = 0; i < connection_count; i++)
	{
		connection_points_vector[i] -= connection_points_vector[i] > neuron_i;
	}
	hipFree(connection_points);
	hipFree(weights);
  hipFree(connection_neuron_i);
	hipDeviceSynchronize();

	hipMalloc(&connection_points, sizeof(size_t) * connection_count);
	hipMalloc(&weights, sizeof(field_t) * connection_count);
  hipMalloc(&connection_neuron_i, sizeof(size_t) * connection_count);
	hipDeviceSynchronize();

	hipMemcpy(connection_points, connection_points_vector.data(), sizeof(size_t) * connection_count, hipMemcpyHostToDevice);
	hipMemcpy(weights, vector_weights.data(), sizeof(field_t) * connection_count, hipMemcpyHostToDevice);
  hipMemcpy(connection_neuron_i, vector_connection_neuron_i.data(), sizeof(size_t) * connection_count, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
}

IConnections* NeatConnections::connections_specific_clone()
{
	NeatConnections* connections = new NeatConnections();
	hipMalloc(&connections->connection_points, sizeof(size_t) * connection_count);
	hipMalloc(&connections->connection_neuron_i, sizeof(size_t) * connection_count);
	hipDeviceSynchronize();
	hipMemcpy(connections->connection_points, connection_points, sizeof(size_t) * connection_count, hipMemcpyDeviceToDevice);
	hipMemcpy(connections->connection_neuron_i, connection_neuron_i, sizeof(size_t) * neuron_count, hipMemcpyDeviceToDevice);
	hipDeviceSynchronize();
	return connections;
}

void specific_save(FILE* file)
{
	size_t *host_connection_points, *host_connection_neuron_i;
	host_connection_points = new size_t[connection_count];
	host_connection_neuron_i = new size_t[connection_count];

	hipMemcpy(host_connection_points, connection_points, sizeof(size_t) * connection_count, hipMemcpyDeviceToHost);
	hipMemcpy(host_connection_neuron_i, connection_neuron_i, sizeof(size_t) * connection_count, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	
	fwrite(host_connection_points, sizeof(size_t), connection_count, file);
	fwrite(host_connection_neuron_i, sizeof(size_t), connection_count, file);

	delete[] host_connection_points;
	delete[] host_connection_neuron_i;
}

void NeatConnections::specific_deallocate()
{
	hipFree(connection_points);
	hipFree(connection_neuron_i);
}

