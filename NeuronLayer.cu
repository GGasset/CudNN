#ifndef NEURONLAYER_DEFINITIONS
#define NEURONLAYER_DEFINITIONS

#include "NeuronLayer.h"

NeuronLayer::NeuronLayer(IConnections* connections, size_t neuron_count, ActivationFunctions activation)
{
	set_neuron_count(neuron_count);
	this->activation = activation;
	execution_values_per_neuron = 1;
	layer_gradient_count = connections->connection_count + neuron_count;

	size_t neuron_gradient_i = 0;
	size_t* host_neuron_gradients_starts = new size_t[neuron_count];
	for (size_t i = 0; i < neuron_count; i++)
	{
		host_neuron_gradients_starts[i] = neuron_gradient_i;

		size_t neuron_connection_count = connections->get_connection_count_at(i);
		neuron_gradient_i += neuron_connection_count + 1;
	}

	hipMalloc(&neuron_gradients_starts, sizeof(size_t) * neuron_count);
	hipDeviceSynchronize();
	
	hipMemcpy(neuron_gradients_starts, host_neuron_gradients_starts, sizeof(size_t) * neuron_count, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
}

void NeuronLayer::layer_specific_deallocate()
{
	hipFree(neuron_gradients_starts);
	if (connection_associated_gradient_counts)
		hipFree(connection_associated_gradient_counts);
}

void NeuronLayer::execute(
	data_t* activations, size_t activations_start,
	data_t* execution_values, size_t execution_values_start
)
{
	connections->linear_function(activations_start, activations,
		execution_values, execution_values_start, execution_values_layer_start, execution_values_per_neuron
	);
	switch (activation)
	{
	case ActivationFunctions::sigmoid:
		sigmoid_activation kernel(neuron_count / 32 + (neuron_count % 32 > 0), 32) (
			activations, activations_start, layer_activations_start, true,
			execution_values, execution_values_start, execution_values_layer_start, execution_values_per_neuron, 0, 0, 0,
			neuron_count
		);
		break;
	case ActivationFunctions::_tanh:
		tanh_activation kernel(neuron_count / 32 + (neuron_count % 32 > 0), 32) (
			activations, activations_start, layer_activations_start, true,
			execution_values, execution_values_start, execution_values_layer_start, execution_values_per_neuron, 0, 0, 0,
			neuron_count
		);
		break;
	default:
		break;
	}
	hipDeviceSynchronize();
}

void NeuronLayer::calculate_gradients(
	data_t* activations, size_t activations_start,
	data_t* execution_values, size_t execution_values_start,
	data_t* derivatives, size_t derivatives_start,
	data_t* gradients, size_t next_gradients_start, size_t gradients_start,
	data_t* costs, size_t costs_start
)
{
	neuron_gradient_calculation kernel(neuron_count / 32 + (neuron_count % 32 > 0), 32) (
		execution_values, execution_values_start, execution_values_layer_start,
		gradients, gradients_start, layer_gradients_start, neuron_gradients_starts,
		costs, costs_start, layer_activations_start,
		activation,
		neuron_count
	);
	hipDeviceSynchronize();
	connections->calculate_gradients(
		activations, activations_start, gradients, gradients_start, layer_gradients_start, neuron_gradients_starts,
		costs, costs_start
	);
	hipDeviceSynchronize();
}

void NeuronLayer::subtract_gradients(data_t* gradients, size_t gradients_start, data_t learning_rate, short* dropout, data_t gradient_clip)
{
	connections->subtract_gradients(
		gradients, gradients_start, layer_gradients_start, neuron_gradients_starts,
		learning_rate, dropout, gradient_clip
	);
}

void NeuronLayer::add_neuron(size_t previous_layer_length, size_t previous_layer_activations_start, float previous_layer_connection_probability, size_t min_connections)
{
	size_t added_connections = connections->connection_count;
	connections->add_neuron(previous_layer_length, previous_layer_activations_start, previous_layer_connection_probability, min_connections);
	added_connections = connections->connection_count - added_connections;


	layer_derivative_count += derivatives_per_neuron;
	layer_gradient_count++;
	layer_gradient_count += added_connections;
	
	size_t* new_neuron_gradients_starts = new size_t[neuron_count + 1];
	hipMemcpy(new_neuron_gradients_starts, neuron_gradients_starts, sizeof(size_t) * neuron_count, hipMemcpyDeviceToHost);
	new_neuron_gradients_starts[neuron_count] = new_neuron_gradients_starts[neuron_count - 1] + added_connections;
	hipDeviceSynchronize();
	
	hipFree(neuron_gradients_starts);
	hipDeviceSynchronize();

	set_neuron_count(neuron_count + 1);

	hipMalloc(&neuron_gradients_starts, sizeof(size_t) * neuron_count);
	hipDeviceSynchronize();
	hipMemcpy(neuron_gradients_starts, new_neuron_gradients_starts, sizeof(size_t) * neuron_count, hipMemcpyHostToDevice);
	delete[] new_neuron_gradients_starts;
	hipDeviceSynchronize();
}

void NeuronLayer::adjust_to_added_neuron(size_t added_neuron_i, float connection_probability)
{
	auto added_connections_neuron_i = std::vector<size_t>();
	connections->adjust_to_added_neuron(added_neuron_i, connection_probability, &added_connections_neuron_i);
	for (size_t i = 0; i < added_connections_neuron_i.size(); i++)
	{
		layer_gradient_count++;
		size_t added_connection_neuron_i = added_connections_neuron_i[i];
		for (size_t j = added_connection_neuron_i + 1; j < neuron_count; j++)
			neuron_gradients_starts[j]++;
	}
}

void NeuronLayer::remove_neuron(size_t layer_neuron_i)
{
	size_t removed_connection_count = connections->connection_count;
	size_t* tmp_neuron_gradients_starts = 0;
	
	connections->remove_neuron(layer_neuron_i);
	removed_connection_count -= connections->connection_count;
	layer_gradient_count -= removed_connection_count;

	set_neuron_count(neuron_count - 1);

	hipMalloc(&tmp_neuron_gradients_starts, sizeof(size_t) * neuron_count);
	hipDeviceSynchronize();
	hipMemcpy(tmp_neuron_gradients_starts, neuron_gradients_starts, sizeof(size_t) * layer_neuron_i, hipMemcpyDeviceToDevice);
	hipMemcpy(tmp_neuron_gradients_starts + layer_neuron_i, neuron_gradients_starts + layer_neuron_i + 1, sizeof(size_t) * (neuron_count - layer_neuron_i), hipMemcpyDeviceToDevice);
	hipDeviceSynchronize();
	hipFree(neuron_gradients_starts);
	hipDeviceSynchronize();
	neuron_gradients_starts = tmp_neuron_gradients_starts;
}

void NeuronLayer::adjust_to_removed_neuron(size_t neuron_i)
{
	auto deleted_connections_neuron_i = std::vector<size_t>();
	connections->adjust_to_removed_neuron(neuron_i, &deleted_connections_neuron_i);
	for (size_t i = 0; i < deleted_connections_neuron_i.size(); i++)
	{
		layer_gradient_count--;
		size_t added_connection_neuron_i = deleted_connections_neuron_i[i];
		for (size_t j = added_connection_neuron_i + 1; j < neuron_count; j++)
			neuron_gradients_starts[j]--;
	}
}

#endif